#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "padding.cuh"

TEST(PaddingTest, SimplePaddingTest) {
    hipError_t cudaStatus;

    int w = 2;
    int h = 3;
    int n = 2;
    int p = 1;

    float* d_input;
    float* d_padded;

    int inputSize  = w * h * n;
    int paddedSize = (w + 2 * p) * (h + 2 * p) * n;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * inputSize);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_padded, sizeof(float) * paddedSize);
    EXPECT_EQ(cudaStatus, hipSuccess);

    /*
    Matrix channel 0:
    0  1
    2  3
    4  5
    Matrix channel 1:
    6  7
    8  9
    10 11

    Represented as a vector:

    0 1 2 3 4 5 6 7 8 9 10 11
    */

    std::vector<float> input = {0.0f, 1.0f, 2.0f,  3.0f, 4.0f, 5.0f,
                                6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f};

    cudaStatus = hipMemcpy(
        d_input, input.data(), sizeof(float) * inputSize, hipMemcpyHostToDevice
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    int THREADS_PER_BLOCK = 64;
    int BLOCKS            = paddedSize / THREADS_PER_BLOCK + 1;

    pad_matrix_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(
        d_input, d_padded, w, h, n, p
    );
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    // clang-format off
    std::vector<float> expectedOutput = {
        // channel 0
        0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 1.0f, 0.0f,
        0.0f, 2.0f, 3.0f, 0.0f,
        0.0f, 4.0f, 5.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f,
        // channel 1
        0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 6.0f, 7.0f, 0.0f,
        0.0f, 8.0f, 9.0f, 0.0f,
        0.0f, 10.0f, 11.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f
    };
    // clang-format on

    std::vector<float> output(paddedSize);

    cudaStatus = hipMemcpy(
        output.data(), d_padded, sizeof(float) * paddedSize,
        hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < paddedSize; i++) {
        EXPECT_NEAR(expectedOutput[i], output[i], 1e-5);
    }


    hipFree(d_input);
    hipFree(d_padded);
}
