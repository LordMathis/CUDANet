#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "padding.cuh"
#include "test_cublas_fixture.cuh"

class PaddingTest : public CublasTestFixture {
  protected:
    hipError_t    cudaStatus;
    hipblasStatus_t hipblasStatus_t;
};

TEST_F(PaddingTest, SimplePaddingTest) {
    int w = 2;
    int h = 3;
    int n = 2;
    int p = 1;

    float* d_input;
    float* d_padded;

    int inputSize  = w * h * n;
    int paddedSize = (w + 2 * p) * (h + 2 * p) * n;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * inputSize);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc(
        (void**)&d_padded, sizeof(float) * paddedSize
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    /*
    Matrix channel 0:
    0  1
    2  3
    4  5
    Matrix channel 1:
    6  7
    8  9
    10 11

    Represented as column major vector:

    0 2 4 1 3 5 6 8 10 7 9 11
    */

    std::vector<float> input = {0.0f, 2.0f, 4.0f,  1.0f, 3.0f, 5.0f,
                                6.0f, 8.0f, 10.0f, 7.0f, 9.0f, 11.0f};

    hipblasStatus_t =
        hipblasSetVector(inputSize, sizeof(float), input.data(), 1, d_input, 1);
    EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

    pad_matrix_kernel<<<1, 1>>>(d_input, d_padded, w, h, n, p);
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> expectedOutput = {
        0.0f, 0.0f, 0.0f, 0.0f,  0.0f, 0.0f, 0.0f, 2.0f, 4.0f,  0.0f,
        0.0f, 1.0f, 3.0f, 5.0f,  0.0f, 0.0f, 0.0f, 0.0f, 0.0f,  0.0f,
        0.0f, 0.0f, 0.0f, 0.0f,  0.0f, 0.0f, 6.0f, 8.0f, 10.0f, 0.0f,
        0.0f, 7.0f, 9.0f, 11.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,  0.0f
    };

    std::vector<float> output(paddedSize);
    hipblasStatus_t = hipblasGetVector(
        paddedSize, sizeof(float), d_padded, 1, output.data(), 1
    );

    std::cout << "Actual output: " << std::endl;
    for (int i = 0; i < paddedSize; i++) {
        std::cout << output[i] << " ";
    }
    std::cout << std::endl;

    for (int i = 0; i < paddedSize; i++) {
        EXPECT_NEAR(expectedOutput[i], output[i], 1e-5);
    }
}