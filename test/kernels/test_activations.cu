#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "activations.cuh"

TEST(ActivationsTest, SigmoidSanityCheck) {

    hipError_t cudaStatus;

    float input[3] = {-100.0f, 0.0f, 100.0f};

    std::vector<float> expected_output = {0.0f, 0.5f, 1.0f};

    float* d_input;
    float* d_output;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * 3);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * 3);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMemcpy(d_input, input, sizeof(float) * 3, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    sigmoid_kernel<<<1, 3>>>(d_input, d_output, 3);
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> output(3);

    cudaStatus = hipMemcpy(output.data(), d_output, sizeof(float) * 3, hipMemcpyDeviceToHost);
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < 3; i++) {
        EXPECT_NEAR(expected_output[i], output[i], 1e-5);
    }

    hipFree(d_input);
    hipFree(d_output);
}