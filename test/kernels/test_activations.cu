#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "activations.cuh"
#include "test_cublas_fixture.cuh"

class ActivationsTest : public CublasTestFixture {
  protected:
    hipError_t    cudaStatus;
    hipblasStatus_t hipblasStatus_t;
};

TEST_F(ActivationsTest, SigmoidSanityCheck) {
    float input[3] = {-100.0f, 0.0f, 100.0f};

    std::vector<float> expected_output = {0.0f, 0.5f, 1.0f};

    float* d_input;
    float* d_output;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * 3);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * 3);
    EXPECT_EQ(cudaStatus, hipSuccess);

    hipblasStatus_t = hipblasSetVector(3, sizeof(float), input, 1, d_input, 1);
    EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

    sigmoid_kernel<<<1, 3>>>(d_input, d_output, 3);
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> output(3);

    hipblasStatus_t =
        hipblasGetVector(3, sizeof(float), d_output, 1, output.data(), 1);
    EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

    for (int i = 0; i < 3; i++) {
        EXPECT_NEAR(expected_output[i], output[i], 1e-5);
    }

    hipFree(d_input);
    hipFree(d_output);
}