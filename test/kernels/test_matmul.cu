#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <vector>

#include "cuda_helper.cuh"
#include "vector.cuh"
#include "matmul.cuh"

TEST(MatMulTest, MatVecMulTest) {
    hipError_t cudaStatus;

    int w = 10;
    int h = 5;

    float* d_matrix;
    float* d_vector;
    float* d_output;

    cudaStatus = hipMalloc((void**)&d_matrix, sizeof(float) * w * h);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_vector, sizeof(float) * w);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * h);
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> matrix = {
        0.643f, 0.912f, 0.723f, 0.587f, 0.155f, 0.932f, 0.391f, 0.279f, 0.846f, 0.788f,
        0.641f, 0.445f, 0.528f, 0.316f, 0.247f, 0.181f, 0.549f, 0.328f, 0.919f, 0.405f,
        0.733f, 0.287f, 0.901f, 0.602f, 0.816f, 0.495f, 0.797f, 0.210f, 0.305f, 0.613f,
        0.178f, 0.856f, 0.724f, 0.263f, 0.559f, 0.677f, 0.193f, 0.389f, 0.488f, 0.848f,
        0.121f, 0.734f, 0.587f, 0.904f, 0.312f, 0.672f, 0.807f, 0.478f, 0.581f, 0.964f
    };
    std::vector<float> vector = {
        0.643f, 0.912f, 0.723f, 0.587f, 0.155f, 0.932f, 0.391f, 0.279f, 0.846f, 0.788f
    };

    cudaStatus = hipMemcpy(d_matrix, matrix.data(), sizeof(float) * w * h, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMemcpy(d_vector, vector.data(), sizeof(float) * w, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    int grid_size = (std::max(w, h) + BLOCK_SIZE - 1) / BLOCK_SIZE;


    CUDANet::Utils::clear(d_output, h);

    CUDANet::Kernels::mat_vec_mul<<<grid_size, BLOCK_SIZE>>>(d_matrix, d_vector, d_output, w, h);
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> output_gpu(h);
    cudaStatus = hipMemcpy(output_gpu.data(), d_output, sizeof(float) * h, hipMemcpyDeviceToHost);
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < h; i++) {
        float sum = 0.0f;
        for (int j = 0; j < w; j++) {
            sum += matrix[i * w + j] * vector[j];
        }
        EXPECT_NEAR(sum, output_gpu[i], 1e-5f);
    }

    hipFree(d_matrix);
    hipFree(d_vector);
    hipFree(d_output);

    
}

TEST(MatMulTest, MaxReduceTest) {
    hipError_t cudaStatus;

    const int n = 1 << 16;

    std::vector<float> input(n);
    for (int i = 0; i < n; i++) {
        input[i] = i;
    }

    float* d_input;
    float* d_output;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * n);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * n);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMemcpy(d_input, input.data(), sizeof(float) * n, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    const int grid_size = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    CUDANet::Kernels::max_reduce<<<grid_size, BLOCK_SIZE>>>(d_input, d_output, n);

    int remaining = grid_size;
    while (remaining > 1) {
        int blocks_needed = (remaining + BLOCK_SIZE - 1) / BLOCK_SIZE;
        CUDANet::Kernels::max_reduce<<<blocks_needed, BLOCK_SIZE>>>(d_output, d_output, remaining);
        remaining = blocks_needed;
    }

    std::vector<float> output(n);
    cudaStatus = hipMemcpy(output.data(), d_output, sizeof(float), hipMemcpyDeviceToHost);
    EXPECT_EQ(cudaStatus, hipSuccess);

    EXPECT_EQ(output[0], 65535.0f);

    hipFree(d_input);
    hipFree(d_output);

    
}

TEST(MatMulTest, VecExpTest) {
    hipError_t cudaStatus;

    float input[6] = {22.496f,  36.9006f, 30.9904f,
                      28.4213f, 26.4541f, 31.7887f};

    std::vector<float> expected = {5886928896.0f,     1.06102872080384e+16f,
                                   28771323215872.0f, 2204012904448.0f,
                                   308226162688.0f,   63922983927808.0f};

    float* d_input;
    float* d_output;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * 6);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * 6);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus =
        hipMemcpy(d_input, input, sizeof(float) * 6, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    CUDANet::Kernels::vec_exp<<<1, 6>>>(d_input, d_output, 6);
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> output(6);

    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * 6, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < 6; i++) {
        EXPECT_NEAR(expected[i], output[i], 1e7f);
    }

    hipFree(d_input);
    hipFree(d_output);

    
}

TEST(MatMulTest, SumReduceTest) {
    hipError_t cudaStatus;

    const int n = 1 << 16;

    std::vector<float> input(n);
    for (int i = 0; i < n; i++) {
        input[i] = 1.0f;
    }

    const float expected = n;

    float* d_input = nullptr;
    float* d_sum = nullptr;

    const int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * n);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_sum, sizeof(float) * n);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus =
        hipMemcpy(d_input, input.data(), sizeof(float) * n, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    CUDANet::Kernels::sum_reduce<<<gridSize, BLOCK_SIZE>>>(
        d_input, d_sum, n
    );

    int remaining = gridSize;
    while (remaining > 1) {
        int blocks_needed = (remaining + BLOCK_SIZE - 1) / BLOCK_SIZE;
        CUDANet::Kernels::sum_reduce<<<blocks_needed, BLOCK_SIZE>>>(d_sum, d_sum, remaining);
        remaining = blocks_needed;
    }


    std::vector<float> sum(n);
    cudaStatus = hipMemcpy(
        sum.data(), d_sum, sizeof(float) * n, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    EXPECT_FLOAT_EQ(expected, sum[0]);    

    hipFree(d_input);
    hipFree(d_sum);

    
}