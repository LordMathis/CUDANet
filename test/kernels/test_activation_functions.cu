#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "activation_functions.cuh"

TEST(ActivationFunctionsTest, SigmoidSanityCheck) {
    hipError_t cudaStatus;

    float input[3] = {-100.0f, 0.0f, 100.0f};

    std::vector<float> expected_output = {0.0f, 0.5f, 1.0f};

    float* d_input;
    float* d_output;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * 3);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * 3);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus =
        hipMemcpy(d_input, input, sizeof(float) * 3, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    CUDANet::Kernels::sigmoid<<<1, 3>>>(d_input, d_output, 3);
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> output(3);

    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * 3, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < 3; i++) {
        EXPECT_NEAR(expected_output[i], output[i], 1e-5);
    }

    hipFree(d_input);
    hipFree(d_output);
}

TEST(ActivationFunctionsTest, SoftmaxExpTest) {
    hipError_t cudaStatus;

    float input[6] = {22.496f,  36.9006f, 30.9904f,
                      28.4213f, 26.4541f, 31.7887f};

    std::vector<float> expected = {5886928896.0f,     1.06102872080384e+16f,
                                   28771323215872.0f, 2204012904448.0f,
                                   308226162688.0f,   63922983927808.0f};

    float* d_input;
    float* d_output;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * 6);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * 6);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus =
        hipMemcpy(d_input, input, sizeof(float) * 6, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    CUDANet::Kernels::softmax_exp<<<1, 6>>>(d_input, d_output, 6);
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> output(6);

    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * 6, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < 6; i++) {
        EXPECT_NEAR(expected[i], output[i], 1e7);
    }

    hipFree(d_input);
    hipFree(d_output);
}

TEST(ActivationFunctionsTest, SoftmaxSumTest) {
    hipError_t cudaStatus;

    std::vector<float> input = {5886928896.0f,     1.06102872080384e+16f,
                                28771323215872.0f, 2204012904448.0f,
                                308226162688.0f,   63922983927808.0f};

    float* d_input;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * 6);
    EXPECT_EQ(cudaStatus, hipSuccess);
}