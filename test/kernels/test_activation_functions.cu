#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "activation_functions.cuh"
#include "cuda_helper.cuh"

TEST(ActivationFunctionsTest, SigmoidSanityCheck) {
    hipError_t cudaStatus;

    float input[3] = {-100.0f, 0.0f, 100.0f};

    std::vector<float> expected_output = {0.0f, 0.5f, 1.0f};

    float* d_input;
    float* d_output;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * 3);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * 3);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus =
        hipMemcpy(d_input, input, sizeof(float) * 3, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    CUDANet::Kernels::sigmoid<<<1, 3>>>(d_input, d_output, 3);
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> output(3);

    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * 3, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < 3; i++) {
        EXPECT_NEAR(expected_output[i], output[i], 1e-5);
    }

    hipFree(d_input);
    hipFree(d_output);
}

TEST(ActivationFunctionsTest, SoftmaxExpTest) {
    hipError_t cudaStatus;

    float input[6] = {22.496f,  36.9006f, 30.9904f,
                      28.4213f, 26.4541f, 31.7887f};

    std::vector<float> expected = {5886928896.0f,     1.06102872080384e+16f,
                                   28771323215872.0f, 2204012904448.0f,
                                   308226162688.0f,   63922983927808.0f};

    float* d_input;
    float* d_output;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * 6);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * 6);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus =
        hipMemcpy(d_input, input, sizeof(float) * 6, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    CUDANet::Kernels::softmax_exp<<<1, 6>>>(d_input, d_output, 6);
    cudaStatus = hipDeviceSynchronize();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> output(6);

    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * 6, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < 6; i++) {
        EXPECT_NEAR(expected[i], output[i], 1e7);
    }

    hipFree(d_input);
    hipFree(d_output);
}

TEST(ActivationFunctionsTest, SoftmaxSumTest) {
    hipError_t cudaStatus;

    const int n = 10;
    std::vector<float> input(n);
    for (int i = 0; i < n; i++) {
        input[i] = i;
    }

    const float expected = n * (n - 1) / 2;

    float* d_input;
    float* d_sum;

    const int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * n);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_sum, sizeof(float) * n);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus =
        hipMemcpy(d_input, input.data(), sizeof(float) * n, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    CUDANet::Kernels::softmax_sum<<<gridSize, BLOCK_SIZE>>>(
        d_input, d_sum
    );

    CUDANet::Kernels::softmax_sum<<<1, BLOCK_SIZE>>>(
        d_sum, d_sum
    );

    CUDANet::Kernels::softmax_sum<<<1, BLOCK_SIZE>>>(
        d_sum, d_sum
    );

    std::vector<float> sum(n);
    cudaStatus = hipMemcpy(
        sum.data(), d_sum, sizeof(float) * n, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    EXPECT_FLOAT_EQ(expected, sum[0]);    
}