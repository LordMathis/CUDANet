#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <vector>

#include "activation.cuh"
#include "batch_norm.cuh"

class BatchNormLayerTest : public ::testing::Test {
  protected:
    shape2d              inputSize;
    int                nChannels;
    std::vector<float> weights;
    std::vector<float> biases;
    std::vector<float> input;
    std::vector<float> expected;

    float                        *d_input;
    float                        *d_output;
    CUDANet::Layers::BatchNorm2d *batchNorm;

    virtual void SetUp() override {
        d_input   = nullptr;
        d_output  = nullptr;
        batchNorm = nullptr;
    }

    virtual void TearDown() override {
        if (d_input) {
            hipFree(d_input);
        }
    }

    void runTest() {
        hipError_t cudaStatus;

        batchNorm = new CUDANet::Layers::BatchNorm2d(
            inputSize, nChannels, 1e-5f, CUDANet::Layers::ActivationType::NONE
        );

        batchNorm->setWeights(weights.data());
        batchNorm->setBiases(biases.data());

        cudaStatus = hipGetLastError();
        EXPECT_EQ(cudaStatus, hipSuccess);

        cudaStatus =
            hipMalloc((void **)&d_input, sizeof(float) * input.size());
        EXPECT_EQ(cudaStatus, hipSuccess);

        cudaStatus = hipMemcpy(
            d_input, input.data(), sizeof(float) * input.size(),
            hipMemcpyHostToDevice
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        d_output = batchNorm->forward(d_input);

        std::vector<float> output(input.size());
        cudaStatus = hipMemcpy(
            output.data(), d_output, sizeof(float) * output.size(),
            hipMemcpyDeviceToHost
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        for (int i = 0; i < output.size(); ++i) {
            EXPECT_NEAR(output[i], expected[i], 1e-5);
        }

        delete batchNorm;
    }
};

TEST_F(BatchNormLayerTest, BatchNormSmallForwardTest) {
    inputSize = {4, 4};
    nChannels = 2;

    weights = {0.63508f, 0.64903f};
    biases  = {0.25079f, 0.66841f};

    // clang-format off
    input = {
        // Channel 0
        0.38899f, 0.80478f, 0.48836f, 0.97381f,
        0.57508f, 0.60835f, 0.65467f, 0.00168f,
        0.65869f, 0.74235f, 0.17928f, 0.70349f,
        0.15524f, 0.38664f, 0.23411f, 0.7137f,
        // Channel 1
        0.32473f, 0.15698f, 0.314f, 0.60888f,
        0.80268f, 0.99766f, 0.93694f, 0.89237f,
        0.13449f, 0.27367f, 0.53036f, 0.18962f,
        0.57672f, 0.48364f, 0.10863f, 0.0571f
    };
    // clang-format on

    expected = {-0.06007f, 0.951f,    0.18157f,  1.36202f, 0.39244f,  0.47335f,
                0.58598f,  -1.00188f, 0.59576f,  0.79919f, -0.57001f, 0.70469f,
                -0.62847f, -0.06578f, -0.43668f, 0.72952f, 0.37726f,  0.02088f,
                0.35446f,  0.98092f,  1.39264f,  1.80686f, 1.67786f,  1.58318f,
                -0.0269f,  0.26878f,  0.81411f,  0.09022f, 0.9126f,   0.71485f,
                -0.08184f, -0.19131f};

    runTest();
}

TEST_F(BatchNormLayerTest, BatchNormNonSquareInputTest) {
    inputSize = {4, 6};  // Non-square input
    nChannels = 2;
    weights   = {0.63508f, 0.64903f};
    biases    = {0.25079f, 0.66841f};

    input = {// Channel 0
             0.38899f, 0.80478f, 0.48836f, 0.97381f, 0.21567f, 0.92312f,
             0.57508f, 0.60835f, 0.65467f, 0.00168f, 0.31567f, 0.71345f,
             0.65869f, 0.74235f, 0.17928f, 0.70349f, 0.12856f, 0.95645f,
             0.15524f, 0.38664f, 0.23411f, 0.7137f, 0.26789f, 0.83412f,
             // Channel 1
             0.32473f, 0.15698f, 0.314f, 0.60888f, 0.23145f, 0.78945f, 0.80268f,
             0.99766f, 0.93694f, 0.89237f, 0.61234f, 0.92314f, 0.13449f,
             0.27367f, 0.53036f, 0.18962f, 0.45623f, 0.14523f, 0.57672f,
             0.48364f, 0.10863f, 0.0571f, 0.78934f, 0.67545f
    };

    expected = {-0.05598f, 0.87495f,  0.1665f,   1.2534f,   -0.44404f,
                1.13991f,  0.36066f,  0.43515f,  0.53886f,  -0.92315f,
                -0.22014f, 0.67047f,  0.54786f,  0.73517f,  -0.52552f,
                0.64817f,  -0.63907f, 1.21453f,  -0.57934f, -0.06124f,
                -0.40275f, 0.67103f,  -0.32712f, 0.94064f,  0.28344f,
                -0.08405f, 0.25993f,  0.90592f,  0.07909f,  1.30149f,
                1.33047f,  1.7576f,   1.62459f,  1.52695f,  0.9135f,
                1.59436f,  -0.13331f, 0.17158f,  0.73391f,  -0.01254f,
                0.57151f,  -0.10979f, 0.83546f,  0.63156f,  -0.18996f,
                -0.30285f, 1.30124f,  1.05175f};

    runTest();
}