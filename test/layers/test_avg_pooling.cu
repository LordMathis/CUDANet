#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <vector>

#include "avg_pooling.cuh"

class AvgPoolingLayerTest : public ::testing::Test {
  protected:
    shape2d              inputSize;
    int                nChannels;
    shape2d              poolingSize;
    shape2d              stride;
    shape2d              padding;
    std::vector<float> input;
    std::vector<float> expected;

    float                         *d_input;
    float                         *d_output;
    CUDANet::Layers::AvgPooling2d *avgPoolingLayer;

    virtual void SetUp() override {
        d_input         = nullptr;
        d_output        = nullptr;
        avgPoolingLayer = nullptr;
    }

    virtual void TearDown() override {
        if (d_input) {
            hipFree(d_input);
        }
    }

    void runTest() {
        hipError_t cudaStatus;

        avgPoolingLayer = new CUDANet::Layers::AvgPooling2d(
            inputSize, nChannels, poolingSize, stride, padding,
            CUDANet::Layers::ActivationType::NONE
        );

        cudaStatus = hipMalloc(
            (void **)&d_input,
            sizeof(float) * inputSize.first * inputSize.second * nChannels
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        cudaStatus = hipMemcpy(
            d_input, input.data(),
            sizeof(float) * inputSize.first * inputSize.second * nChannels,
            hipMemcpyHostToDevice
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        d_output = avgPoolingLayer->forward(d_input);

        int outputSize = avgPoolingLayer->getOutputSize();

        std::vector<float> output(outputSize);
        cudaStatus = hipMemcpy(
            output.data(), d_output, sizeof(float) * outputSize,
            hipMemcpyDeviceToHost
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        for (int i = 0; i < output.size(); ++i) {
            EXPECT_NEAR(expected[i], output[i], 1e-4);
        }

        delete avgPoolingLayer;
    }
};

TEST_F(AvgPoolingLayerTest, AvgPoolForwardTest) {
    inputSize   = {4, 4};
    nChannels   = 2;
    poolingSize = {2, 2};
    stride      = {2, 2};
    padding     = {0, 0};

    input = {
        // clang-format off
        // Channel 0
        0.573f, 0.619f, 0.732f, 0.055f,
        0.243f, 0.316f, 0.573f, 0.619f,
        0.712f, 0.055f, 0.243f, 0.316f,
        0.573f, 0.619f, 0.742f, 0.055f,
        // Channel 1
        0.473f, 0.919f, 0.107f, 0.073f,
        0.073f, 0.362f, 0.973f, 0.059f,
        0.473f, 0.455f, 0.283f, 0.416f,
        0.532f, 0.819f, 0.732f, 0.850f
        // clang-format on
    };

    expected = {0.43775f, 0.49475f, 0.48975f, 0.339f,
                0.45675f, 0.303f,   0.56975f, 0.57025f};

    runTest();
}

TEST_F(AvgPoolingLayerTest, AvgPoolForwardNonSquareInputTest) {
    inputSize   = {4, 6};  // Non-square input
    nChannels   = 2;
    poolingSize = {2, 2};
    stride      = {2, 2};
    padding     = {0, 0};

    input = {// Channel 0
             0.573f, 0.619f, 0.732f, 0.055f, 0.123f, 0.234f, 0.243f, 0.316f,
             0.573f, 0.619f, 0.456f, 0.789f, 0.712f, 0.055f, 0.243f, 0.316f,
             0.654f, 0.987f, 0.573f, 0.619f, 0.742f, 0.055f, 0.321f, 0.654f,
             // Channel 1
             0.473f, 0.919f, 0.107f, 0.073f, 0.321f, 0.654f, 0.073f, 0.362f,
             0.973f, 0.059f, 0.654f, 0.987f, 0.473f, 0.455f, 0.283f, 0.416f,
             0.789f, 0.123f, 0.532f, 0.819f, 0.732f, 0.850f, 0.987f, 0.321f
    };

    expected = {0.43775f, 0.49475f, 0.4005f, 0.48975f, 0.339f,   0.654f,
                0.45675f, 0.303f,   0.654f,  0.56975f, 0.57025f, 0.555f};

    runTest();
}

TEST_F(AvgPoolingLayerTest, AvgPoolForwardNonSquarePoolingTest) {
    inputSize   = {4, 4};
    nChannels   = 2;
    poolingSize = {2, 3};  // Non-square pooling
    stride      = {2, 2};
    padding     = {0, 0};

    input = {// Channel 0
             0.573f, 0.619f, 0.732f, 0.055f, 0.243f, 0.316f, 0.573f, 0.619f,
             0.712f, 0.055f, 0.243f, 0.316f, 0.573f, 0.619f, 0.742f, 0.055f,
             // Channel 1
             0.473f, 0.919f, 0.107f, 0.073f, 0.073f, 0.362f, 0.973f, 0.059f,
             0.473f, 0.455f, 0.283f, 0.416f, 0.532f, 0.819f, 0.732f, 0.850f
    };

    expected = {0.50933f, 0.49067f, 0.4845f, 0.549f};

    runTest();
}

TEST_F(AvgPoolingLayerTest, AvgPoolForwardNonSquareStrideTest) {
    inputSize   = {4, 4};
    nChannels   = 2;
    poolingSize = {2, 2};
    stride      = {1, 2};  // Non-square stride
    padding     = {0, 0};

    input = {// Channel 0
             0.573f, 0.619f, 0.732f, 0.055f, 0.243f, 0.316f, 0.573f, 0.619f,
             0.712f, 0.055f, 0.243f, 0.316f, 0.573f, 0.619f, 0.742f, 0.055f,
             // Channel 1
             0.473f, 0.919f, 0.107f, 0.073f, 0.073f, 0.362f, 0.973f, 0.059f,
             0.473f, 0.455f, 0.283f, 0.416f, 0.532f, 0.819f, 0.732f, 0.850f
    };

    expected = {0.43775f, 0.49475f, 0.3315f,  0.43775f, 0.48975f, 0.339f,
                0.45675f, 0.303f,   0.34075f, 0.43275f, 0.56975f, 0.57025f};

    runTest();
}

TEST_F(AvgPoolingLayerTest, AvgPoolForwardNonSquarePaddingTest) {
    inputSize   = {4, 4};
    nChannels   = 2;
    poolingSize = {2, 2};
    stride      = {2, 2};
    padding     = {1, 0};  // Non-square padding

    input = {// Channel 0
             0.573f, 0.619f, 0.732f, 0.055f, 0.243f, 0.316f, 0.573f, 0.619f,
             0.712f, 0.055f, 0.243f, 0.316f, 0.573f, 0.619f, 0.742f, 0.055f,
             // Channel 1
             0.473f, 0.919f, 0.107f, 0.073f, 0.073f, 0.362f, 0.973f, 0.059f,
             0.473f, 0.455f, 0.283f, 0.416f, 0.532f, 0.819f, 0.732f, 0.850f
    };

    expected = {0.298f, 0.19675f, 0.3315f,  0.43775f, 0.298f,   0.19925f,
                0.348f, 0.045f,   0.34075f, 0.43275f, 0.33775f, 0.3955f};

    runTest();
}