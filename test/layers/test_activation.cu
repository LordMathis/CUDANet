#include "activation.cuh"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>

TEST(ActivationTest, SoftmaxTest) {
    CUDANet::Layers::Activation activation(
        CUDANet::Layers::ActivationType::SOFTMAX, 5
    );

    std::vector<float> input = {0.573f, 0.619f, 0.732f, 0.055f, 0.243f};

    float* d_input;
    hipMalloc((void**)&d_input, sizeof(float) * 5);
    hipMemcpy(d_input, input.data(), sizeof(float) * 5, hipMemcpyHostToDevice);

    activation.activate(d_input);
    std::vector<float> output(5);
    hipMemcpy(
        output.data(), d_input, sizeof(float) * 5, hipMemcpyDeviceToHost
    );
    
    float sum = 0.0f;

    std::vector<float> expected = {0.22055f, 0.23094f, 0.25856f, 0.13139f, 0.15856f};
    for (int i = 0; i < 5; ++i) {
        sum += output[i];
        EXPECT_NEAR(output[i], expected[i], 1e-5f);
    }

    EXPECT_NEAR(sum, 1.0f, 1e-5f);

    hipFree(d_input);
}