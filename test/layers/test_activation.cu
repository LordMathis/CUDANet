#include "activation.cuh"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>


TEST(ActivationTest, SoftmaxTest1) {
    const int inputSize = 5;
    hipError_t cudaStatus;

    CUDANet::Layers::Activation activation(
        CUDANet::Layers::ActivationType::SOFTMAX, inputSize
    );

    std::vector<float> input = {0.573f, 0.619f, 0.732f, 0.055f, 0.243f};

    float* d_input;
    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * inputSize);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMemcpy(d_input, input.data(), sizeof(float) * inputSize, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    activation.activate(d_input);
    std::vector<float> output(5);
    cudaStatus = hipMemcpy(
        output.data(), d_input, sizeof(float) * inputSize, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);
    
    float sum = 0.0f;

    std::vector<float> expected = {0.22055f, 0.23094f, 0.25856f, 0.13139f, 0.15856f};
    for (int i = 0; i < inputSize; ++i) {
        sum += output[i];
        EXPECT_NEAR(output[i], expected[i], 1e-5f);
    }

    EXPECT_NEAR(sum, 1.0f, 1e-5f);

    cudaStatus = hipFree(d_input);
    EXPECT_EQ(cudaStatus, hipSuccess);
}

TEST(ActivationTest, SoftmaxTest2) {
    const int inputSize = 6;
    hipError_t cudaStatus;

    CUDANet::Layers::Activation activation(
        CUDANet::Layers::ActivationType::SOFTMAX, inputSize
    );

    cudaStatus = hipGetLastError();
    EXPECT_EQ(cudaStatus, hipSuccess);

    std::vector<float> input = {22.496f, 36.9006f, 30.9904f, 28.4213f, 26.4541f, 31.7887f};

    float* d_input;
    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * inputSize);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMemcpy(d_input, input.data(), sizeof(float) * inputSize, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    activation.activate(d_input);
    std::vector<float> output(inputSize);
    cudaStatus = hipMemcpy(
        output.data(), d_input, sizeof(float) * inputSize, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);
    
    float sum = 0.0f;

    std::vector<float> expected = {0.0f, 0.99111f, 0.00269f, 0.00021f, 3e-05f, 0.00597f};
    for (int i = 0; i < inputSize; ++i) {
        sum += output[i];
        EXPECT_NEAR(output[i], expected[i], 1e-5f);
    }
    EXPECT_NEAR(sum, 1.0f, 1e-5f);

    // Cleanup
    cudaStatus = hipFree(d_input);
    EXPECT_EQ(cudaStatus, hipSuccess);
}