#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "activation.cuh"
#include "dense.cuh"

class DenseLayerTest : public ::testing::Test {
  protected:
    CUDANet::Layers::Dense commonTestSetup(
        int                         inputSize,
        int                         outputSize,
        std::vector<float>&         input,
        float*                      weights,
        float*                      biases,
        float*&                     d_input,
        CUDANet::Layers::ActivationType activationType
    ) {
        // Create Dense layer
        CUDANet::Layers::Dense denseLayer(inputSize, outputSize, activationType);

        // Set weights and biases
        denseLayer.setWeights(weights);
        denseLayer.setBiases(biases);

        // Allocate device memory
        cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * input.size());
        EXPECT_EQ(cudaStatus, hipSuccess);

        // Copy input to device
        cudaStatus = hipMemcpy(
            d_input, input.data(), sizeof(float) * input.size(),
            hipMemcpyHostToDevice
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        return denseLayer;
    }

    void commonTestTeardown(float* d_input) {
        // Free device memory
        hipFree(d_input);
    }

    hipError_t cudaStatus;
};

TEST_F(DenseLayerTest, Init) {
    for (int i = 1; i < 100; ++i) {
        for (int j = 1; j < 100; ++j) {
            int inputSize  = i;
            int outputSize = j;

            CUDANet::Layers::Dense denseLayer(
                inputSize, outputSize, CUDANet::Layers::ActivationType::SIGMOID
            );
        }
    }
}

TEST_F(DenseLayerTest, setWeights) {
    int inputSize  = 4;
    int outputSize = 5;

    // clang-format off
    std::vector<float> weights = {
        0.5f, 1.0f, 0.2f, 0.8f,
        1.2f, 0.3f, 1.5f, 0.4f,
        0.7f, 1.8f, 0.9f, 0.1f,
        0.4f, 2.0f, 0.6f, 1.1f,
        1.3f, 0.5f, 0.0f, 1.7f
    };
    // clang-format on

    CUDANet::Layers::Dense denseLayer(
        inputSize, outputSize, CUDANet::Layers::ActivationType::SIGMOID
    );

    denseLayer.setWeights(weights.data());
}

TEST_F(DenseLayerTest, ForwardUnitWeightMatrixLinear) {
    int inputSize  = 3;
    int outputSize = 3;

    std::vector<float> input = {1.0f, 2.0f, 3.0f};

    std::vector<float> weights(outputSize * inputSize, 0.0f);
    for (int i = 0; i < inputSize; ++i) {
        for (int j = 0; j < outputSize; ++j) {
            if (i == j) {
                weights[i * outputSize + j] = 1.0f;
            }
        }
    }
    std::vector<float> biases(outputSize, 1.0f);

    float* d_input;
    float* d_output;

    CUDANet::Layers::Dense denseLayer = commonTestSetup(
        inputSize, outputSize, input, weights.data(), biases.data(), d_input,
        CUDANet::Layers::ActivationType::NONE
    );
    d_output = denseLayer.forward(d_input);

    std::vector<float> output(outputSize);
    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * outputSize,
        hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    // Check if the output is a zero vector
    EXPECT_FLOAT_EQ(output[0], 2.0f);
    EXPECT_FLOAT_EQ(output[1], 3.0f);
    EXPECT_FLOAT_EQ(output[2], 4.0f);

    commonTestTeardown(d_input);
}

TEST_F(DenseLayerTest, ForwardRandomWeightMatrixRelu) {
    int inputSize  = 5;
    int outputSize = 4;

    std::vector<float> input = {1.0f, 2.0f, 3.0f, 4.0f, -5.0f};

    // clang-format off
    std::vector<float> weights = {
        0.5f, 1.2f, 0.7f, 0.4f,
        1.3f, 1.0f, 0.3f, 1.8f,
        2.0f, 0.5f, 0.2f, 1.5f,
        0.9f, 0.6f, 0.0f, 0.8f,
        0.4f, 0.1f, 1.1f, 1.7f
    };
    std::vector<float> biases = {0.2f, 0.5f, 0.7f, -1.1f};
    // clang-format on

    float* d_input;
    float* d_output;

    CUDANet::Layers::Dense denseLayer = commonTestSetup(
        inputSize, outputSize, input, weights.data(), biases.data(), d_input,
        CUDANet::Layers::ActivationType::RELU
    );

    d_output = denseLayer.forward(d_input);

    std::vector<float> output(outputSize);
    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * outputSize,
        hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    // weights * inputs = 0.1, 12.5, 8.3, -2.2
    // + biases = 0.3, 13, 9, -3.3

    std::vector<float> expectedOutput = {0.3f, 13.0f, 9.0f, 0.0f};
    for (int i = 0; i < outputSize; ++i) {
        EXPECT_NEAR(
            output[i], expectedOutput[i], 1e-4
        );  // Allow small tolerance for floating-point comparison
    }

    commonTestTeardown(d_input);
}

TEST_F(DenseLayerTest, ForwardRandomWeightMatrixSigmoid) {
    int inputSize  = 5;
    int outputSize = 4;

    // clang-format off
    std::vector<float> input = {0.1f, 0.2f, 0.3f, 0.4f, 0.5f};
    std::vector<float> weights = {
        0.8f, 0.7f, 0.7f, 0.3f, 0.8f,
        0.1f, 0.4f, 0.8f, 0.0f, 0.2f,
        0.2f, 0.5f, 0.7f, 0.3f, 0.0f,
        0.1f, 0.7f, 0.6f, 1.0f, 0.4f
    };
    std::vector<float> biases = {0.1f, 0.2f, 0.3f, 0.4f};
    // clang-format on

    float* d_input;
    float* d_output;

    CUDANet::Layers::Dense denseLayer = commonTestSetup(
        inputSize, outputSize, input, weights.data(), biases.data(), d_input,
        CUDANet::Layers::ActivationType::SIGMOID
    );

    d_output = denseLayer.forward(d_input);

    std::vector<float> output(outputSize);
    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * outputSize,
        hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    // weights * input = 0.95, 0.43, 0.45, 0.93
    // + biases = 1.05, 0.63, 0.75, 1.33
    // sigmoid = 0.740775, 0.652489, 0.679179, 0.790841

    std::vector<float> expectedOutput = {
        0.740775f, 0.652489f, 0.679179f, 0.790841f
    };

    for (int i = 0; i < outputSize; ++i) {
        EXPECT_NEAR(output[i], expectedOutput[i], 1e-5);
    }

    commonTestTeardown(d_input);
}
