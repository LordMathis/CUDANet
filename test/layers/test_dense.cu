#include "gtest/gtest.h"
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include "dense.cuh"
#include "test_cublas_fixture.cuh"

class DenseLayerTest : public CublasTestFixture {
protected:
    Layers::Dense commonTestSetup(int inputSize, int outputSize, std::vector<float>& input, std::vector<std::vector<float>>& weights, std::vector<float>& biases, float*& d_input, float*& d_output) {
        // Create Dense layer
        Layers::Dense denseLayer(inputSize, outputSize, cublasHandle);

        // Set weights and biases
        denseLayer.setWeights(weights);
        denseLayer.setBiases(biases);

        // Allocate device memory
        cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * input.size());
        EXPECT_EQ(cudaStatus, hipSuccess);

        cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * outputSize);
        EXPECT_EQ(cudaStatus, hipSuccess);

        // Copy input to device
        hipblasStatus_t = hipblasSetVector(input.size(), sizeof(float), input.data(), 1, d_input, 1);
        EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

        return denseLayer;
    }

    void commonTestTeardown(float* d_input, float* d_output) {
        // Free device memory
        hipFree(d_input);
        hipFree(d_output);
    }

    hipError_t cudaStatus;
    hipblasStatus_t hipblasStatus_t;
};

TEST_F(DenseLayerTest, Init) {

    for (int i = 1; i < 100; ++i) {
        for (int j = 1; j < 100; ++j) {

            int inputSize = i;
            int outputSize = j;

            // std::cout << "Dense layer: input size = " << inputSize << ", output size = " << outputSize << std::endl;
            Layers::Dense denseLayer(inputSize, outputSize, cublasHandle);
        }    
    }
}

TEST_F(DenseLayerTest, setWeights) {


    int inputSize = 4;
    int outputSize = 5;

    std::vector<std::vector<float>> weights = {
        {0.5f, 1.0f, 0.2f, 0.8f},
        {1.2f, 0.3f, 1.5f, 0.4f},
        {0.7f, 1.8f, 0.9f, 0.1f},
        {0.4f, 2.0f, 0.6f, 1.1f},
        {1.3f, 0.5f, 0.0f, 1.7f}
    };

    Layers::Dense denseLayer(inputSize, outputSize, cublasHandle);

    denseLayer.setWeights(weights);

}

TEST_F(DenseLayerTest, ForwardUnitWeightMatrix) {

    int inputSize = 3;
    int outputSize = 3;

    std::vector<float> input = {1.0f, 2.0f, 3.0f};

    std::vector<std::vector<float>> weights(inputSize, std::vector<float>(outputSize, 0.0f));
    for (int i = 0; i < inputSize; ++i) {
        for (int j = 0; j < outputSize; ++j) {
            if (i == j) {
                weights[i][j] = 1.0f;
            }
        }
    }
    std::vector<float> biases(outputSize, 1.0f);

    float* d_input;
    float* d_output;

    Layers::Dense denseLayer = commonTestSetup(inputSize, outputSize, input, weights, biases, d_input, d_output);
    denseLayer.forward(d_input, d_output);

    std::vector<float> output(outputSize);
    hipblasStatus_t = hipblasGetVector(outputSize, sizeof(float), d_output, 1, output.data(), 1);
    EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

    // Check if the output is a zero vector
    EXPECT_FLOAT_EQ(output[0], 2.0f);
    EXPECT_FLOAT_EQ(output[1], 3.0f);
    EXPECT_FLOAT_EQ(output[2], 4.0f);

    commonTestTeardown(d_input, d_output);
}

TEST_F(DenseLayerTest, ForwardRandomWeightMatrix) {
    int inputSize = 5;
    int outputSize = 4;

    std::vector<float> input = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};

    std::vector<std::vector<float>> weights = {
        {0.5f, 1.2f, 0.7f, 0.4f, 1.3f},
        {1.0f, 0.3f, 1.8f, 2.0f, 0.5f},
        {0.2f, 1.5f, 0.9f, 0.6f, 0.0f},
        {0.8f, 0.4f, 0.1f, 1.1f, 1.7f}
    };    
    std::vector<float> biases = {0.2f, 0.5f, 0.7f, 1.1f};

    float* d_input;
    float* d_output;  

    Layers::Dense denseLayer = commonTestSetup(inputSize, outputSize, input, weights, biases, d_input, d_output);
    
    denseLayer.forward(d_input, d_output);

    std::vector<float> output(outputSize);
    hipblasStatus_t = hipblasGetVector(outputSize, sizeof(float), d_output, 1, output.data(), 1);
    EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

    std::vector<float> expectedOutput = {10.4f, 13.0f, 8.9f, 9.3f};
    for (int i = 0; i < outputSize; ++i) {
        EXPECT_NEAR(output[i], expectedOutput[i], 1e-4); // Allow small tolerance for floating-point comparison
    }

    commonTestTeardown(d_input, d_output);
}
