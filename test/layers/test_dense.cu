#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include <iostream>

#include "activations.cuh"
#include "dense.cuh"
#include "gtest/gtest.h"
#include "test_cublas_fixture.cuh"

class DenseLayerTest : public CublasTestFixture {
  protected:
    Layers::Dense commonTestSetup(
        int                              inputSize,
        int                              outputSize,
        std::vector<float>&              input,
        std::vector<std::vector<float>>& weights,
        std::vector<float>&              biases,
        float*&                          d_input,
        float*&                          d_output,
        std::string                      activation
    ) {
        // Create Dense layer
        Layers::Dense denseLayer(
            inputSize, outputSize, activation, cublasHandle
        );

        // Set weights and biases
        denseLayer.setWeights(weights);
        denseLayer.setBiases(biases);

        // Allocate device memory
        cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * input.size());
        EXPECT_EQ(cudaStatus, hipSuccess);

        cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * outputSize);
        EXPECT_EQ(cudaStatus, hipSuccess);

        // Copy input to device
        hipblasStatus_t = hipblasSetVector(
            input.size(), sizeof(float), input.data(), 1, d_input, 1
        );
        EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

        return denseLayer;
    }

    void commonTestTeardown(float* d_input, float* d_output) {
        // Free device memory
        hipFree(d_input);
        hipFree(d_output);
    }

    hipError_t    cudaStatus;
    hipblasStatus_t hipblasStatus_t;
};

TEST_F(DenseLayerTest, Init) {
    for (int i = 1; i < 100; ++i) {
        for (int j = 1; j < 100; ++j) {
            int inputSize  = i;
            int outputSize = j;

            // std::cout << "Dense layer: input size = " << inputSize << ",
            // output size = " << outputSize << std::endl;
            Layers::Dense denseLayer(
                inputSize, outputSize, "sigmoid", cublasHandle
            );
        }
    }
}

TEST_F(DenseLayerTest, setWeights) {
    int inputSize  = 4;
    int outputSize = 5;

    std::vector<std::vector<float>> weights = {
        {0.5f, 1.0f, 0.2f, 0.8f},
        {1.2f, 0.3f, 1.5f, 0.4f},
        {0.7f, 1.8f, 0.9f, 0.1f},
        {0.4f, 2.0f, 0.6f, 1.1f},
        {1.3f, 0.5f, 0.0f, 1.7f}
    };

    Layers::Dense denseLayer(inputSize, outputSize, "sigmoid", cublasHandle);

    denseLayer.setWeights(weights);
}

TEST_F(DenseLayerTest, ForwardUnitWeightMatrixLinear) {
    int inputSize  = 3;
    int outputSize = 3;

    std::vector<float> input = {1.0f, 2.0f, 3.0f};

    std::vector<std::vector<float>> weights(
        inputSize, std::vector<float>(outputSize, 0.0f)
    );
    for (int i = 0; i < inputSize; ++i) {
        for (int j = 0; j < outputSize; ++j) {
            if (i == j) {
                weights[i][j] = 1.0f;
            }
        }
    }
    std::vector<float> biases(outputSize, 1.0f);

    float* d_input;
    float* d_output;

    Layers::Dense denseLayer = commonTestSetup(
        inputSize, outputSize, input, weights, biases, d_input, d_output,
        "linear"
    );
    denseLayer.forward(d_input, d_output);

    std::vector<float> output(outputSize);
    hipblasStatus_t = hipblasGetVector(
        outputSize, sizeof(float), d_output, 1, output.data(), 1
    );
    EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

    // Check if the output is a zero vector
    EXPECT_FLOAT_EQ(output[0], 2.0f);
    EXPECT_FLOAT_EQ(output[1], 3.0f);
    EXPECT_FLOAT_EQ(output[2], 4.0f);

    commonTestTeardown(d_input, d_output);
}

TEST_F(DenseLayerTest, ForwardRandomWeightMatrixRelu) {
    int inputSize  = 5;
    int outputSize = 4;

    std::vector<float> input = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};

    std::vector<std::vector<float>> weights = {
        {0.5f, 1.2f, 0.7f, 0.4f, 1.3f},
        {1.0f, 0.3f, 1.8f, 2.0f, 0.5f},
        {0.2f, 1.5f, 0.9f, 0.6f, 0.0f},
        {0.8f, 0.4f, 0.1f, 1.1f, 1.7f}
    };
    std::vector<float> biases = {0.2f, 0.5f, 0.7f, 1.1f};

    float* d_input;
    float* d_output;

    Layers::Dense denseLayer = commonTestSetup(
        inputSize, outputSize, input, weights, biases, d_input, d_output, "relu"
    );

    denseLayer.forward(d_input, d_output);

    std::vector<float> output(outputSize);
    hipblasStatus_t = hipblasGetVector(
        outputSize, sizeof(float), d_output, 1, output.data(), 1
    );
    EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

    // weights * inputs = 13.1, 17.5, 8.3, 14.8
    // + biases = 13.3, 18, 9, 15.9

    std::vector<float> expectedOutput = {13.3f, 18.0f, 9.0f, 15.9f};
    for (int i = 0; i < outputSize; ++i) {
        EXPECT_NEAR(
            output[i], expectedOutput[i], 1e-4
        );  // Allow small tolerance for floating-point comparison
    }

    commonTestTeardown(d_input, d_output);
}

TEST_F(DenseLayerTest, ForwardRandomWeightMatrixSigmoid) {
    int inputSize  = 5;
    int outputSize = 4;

    std::vector<float> input = {0.1f, 0.2f, 0.3f, 0.4f, 0.5f};

    std::vector<std::vector<float>> weights = {
        {0.8f, 0.7f, 0.7f, 0.3f, 0.8f},
        {0.1f, 0.4f, 0.8f, 0.0f, 0.2f},
        {0.2f, 0.5f, 0.7f, 0.3f, 0.0f},
        {0.1f, 0.7f, 0.6f, 1.0f, 0.4f}
    };
    std::vector<float> biases = {0.1f, 0.2f, 0.3f, 0.4f};

    float* d_input;
    float* d_output;

    Layers::Dense denseLayer = commonTestSetup(
        inputSize, outputSize, input, weights, biases, d_input, d_output,
        "sigmoid"
    );

    denseLayer.forward(d_input, d_output);

    std::vector<float> output(outputSize);
    hipblasStatus_t = hipblasGetVector(
        outputSize, sizeof(float), d_output, 1, output.data(), 1
    );
    EXPECT_EQ(hipblasStatus_t, HIPBLAS_STATUS_SUCCESS);

    // weights * input = 0.95, 0.43, 0.45, 0.93
    // + biases = 1.05, 0.63, 0.75, 1.33
    // sigmoid = 0.740775, 0.652489, 0.679179, 0.790841

    std::vector<float> expectedOutput = {
        0.740775f, 0.652489f, 0.679179f, 0.790841f
    };

    for (int i = 0; i < outputSize; ++i) {
        EXPECT_NEAR(
            output[i], expectedOutput[i], 1e-5
        );
    }

    commonTestTeardown(d_input, d_output);
}
