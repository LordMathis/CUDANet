#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include "output.cuh"

TEST(OutputLayerTest, OutputForward) {
    hipError_t cudaStatus;

    std::vector<float> input = {0.573f, 0.619f, 0.732f, 0.055f, 0.243f, 0.316f};
    float*             d_input;
    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * 6);
    EXPECT_EQ(cudaStatus, hipSuccess);
    cudaStatus = hipMemcpy(
        d_input, input.data(), sizeof(float) * 6, hipMemcpyHostToDevice
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    CUDANet::Layers::Output outputLayer(6);
    float* h_output = outputLayer.forward(d_input);

    for (int i = 0; i < 6; ++i) {
        EXPECT_EQ(input[i], h_output[i]);
    }

    hipFree(d_input);
    
}