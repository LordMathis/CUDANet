#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "conv2d.cuh"

class Conv2dTest : public ::testing::Test {
  protected:
    CUDANet::Layers::Conv2d commonTestSetup(
        int                         inputSize,
        int                         inputChannels,
        int                         kernelSize,
        int                         stride,
        CUDANet::Layers::Padding    padding,
        int                         numFilters,
        CUDANet::Layers::Activation activation,
        std::vector<float>&         input,
        float*                      kernels,
        float*&                     d_input
    ) {
        // Create Conv2d layer
        CUDANet::Layers::Conv2d conv2d(
            inputSize, inputChannels, kernelSize, stride, padding, numFilters,
            activation
        );

        conv2d.setWeights(kernels);

        // Allocate device memory
        cudaStatus = hipMalloc(
            (void**)&d_input,
            sizeof(float) * inputSize * inputSize * inputChannels
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        // // Copy input to device
        cudaStatus = hipMemcpy(
            d_input, input.data(), sizeof(float) * input.size(),
            hipMemcpyHostToDevice
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        return conv2d;
    }

    void commonTestTeardown(float* d_input) {
        // Free device memory
        hipFree(d_input);
    }

    hipError_t cudaStatus;
};

TEST_F(Conv2dTest, SimpleTest) {
    int                         inputSize     = 4;
    int                         inputChannels = 1;
    int                         kernelSize    = 2;
    int                         stride        = 1;
    CUDANet::Layers::Padding    padding       = CUDANet::Layers::Padding::VALID;
    int                         numFilters    = 1;
    CUDANet::Layers::Activation activation = CUDANet::Layers::Activation::NONE;

    std::vector<float> input   = {1.0f,  2.0f,  3.0f,  4.0f,  5.0f,  6.0f,
                                  7.0f,  8.0f,  9.0f,  10.0f, 11.0f, 12.0f,
                                  13.0f, 14.0f, 15.0f, 16.0f};
    std::vector<float> kernels = {
        1.0f,
        2.0f,
        3.0f,
        4.0f,
    };

    float* d_input;
    float* d_output;

    CUDANet::Layers::Conv2d conv2d = commonTestSetup(
        inputSize, inputChannels, kernelSize, stride, padding, numFilters,
        activation, input, kernels.data(), d_input
    );

    int outputSize = (inputSize - kernelSize) / stride + 1;
    EXPECT_EQ(outputSize, conv2d.getOutputSize());

    d_output = conv2d.forward(d_input);

    std::vector<float> expected = {44.0f,  54.0f,  64.0f,  84.0f, 94.0f,
                                   104.0f, 124.0f, 134.0f, 144.0f};
    std::vector<float> output(outputSize * outputSize * numFilters);

    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * output.size(),
        hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < output.size(); ++i) {
        EXPECT_FLOAT_EQ(expected[i], output[i]);
    }

    commonTestTeardown(d_input);
}

TEST_F(Conv2dTest, PaddedTest) {
    int                         inputSize     = 5;
    int                         inputChannels = 3;
    int                         kernelSize    = 3;
    int                         stride        = 1;
    CUDANet::Layers::Padding    padding       = CUDANet::Layers::Padding::SAME;
    int                         numFilters    = 2;
    CUDANet::Layers::Activation activation = CUDANet::Layers::Activation::NONE;

    // clang-format off
    std::vector<float> input = {
        // Channel 1
        0.823f, 0.217f, 0.435f, 0.981f, 0.742f,
        0.109f, 0.518f, 0.374f, 0.681f, 0.147f,
        0.956f, 0.729f, 0.654f, 0.087f, 0.392f,
        0.784f, 0.921f, 0.543f, 0.231f, 0.816f,
        0.472f, 0.614f, 0.102f, 0.987f, 0.398f,
        // Channel 2
        0.051f, 0.756f, 0.841f, 0.293f, 0.128f,
        0.417f, 0.632f, 0.095f, 0.184f, 0.529f,
        0.871f, 0.958f, 0.213f, 0.347f, 0.725f,
        0.461f, 0.012f, 0.278f, 0.195f, 0.649f,
        0.853f, 0.707f, 0.988f, 0.988f, 0.322f,
        // Channel 3
        0.345f, 0.123f, 0.789f, 0.123f, 0.456f,
        0.456f, 0.789f, 0.123f, 0.345f, 0.123f,
        0.789f, 0.123f, 0.345f, 0.123f, 0.456f,
        0.123f, 0.345f, 0.123f, 0.789f, 0.123f,
        0.345f, 0.123f, 0.789f, 0.123f, 0.456f
    };

    std::vector<float> kernels = {
        // Filter 1, Channel 1
        0.128f, 0.754f, 0.987f,
        0.321f, 0.412f, 0.635f,
        0.298f, 0.017f, 0.845f,
        // Filter 1, Channel 2
        0.514f, 0.729f, 0.952f,
        0.684f, 0.378f, 0.159f,
        0.823f, 0.547f, 0.216f,
        // Filter 1, Channel 3
        0.983f, 0.231f, 0.456f,
        0.178f,  0.654f, 0.821f,
        0.345f, 0.987f, 0.123f,
        // Filter 2, Channel 1
        0.789f, 0.543f, 0.210f,
        0.012f, 0.371f, 0.638f,
        0.456f, 0.198f, 0.907f,
        // Filter 2, Channel 2
        0.101f, 0.432f, 0.759f,
        0.234f, 0.567f, 0.890f,
        0.543f, 0.876f, 0.219f,
        // Filter 2, Channel 3
        0.345f, 0.678f, 0.011f,
        0.678f, 0.011f, 0.345f,
        0.011f, 0.345f, 0.678f
    };
    // clang-format on

    float* d_input;
    float* d_output;

    CUDANet::Layers::Conv2d conv2d = commonTestSetup(
        inputSize, inputChannels, kernelSize, stride, padding, numFilters,
        activation, input, kernels.data(), d_input
    );

    EXPECT_EQ(inputSize, conv2d.getOutputSize());

    d_output = conv2d.forward(d_input);

    std::vector<float> output(
        conv2d.getOutputSize() * conv2d.getOutputSize() * numFilters
    );
    hipMemcpy(
        output.data(), d_output,
        sizeof(float) * conv2d.getOutputSize() * conv2d.getOutputSize() *
            numFilters,
        hipMemcpyDeviceToHost
    );

    // Generated by tools/generate_conv2d_test.py
    std::vector<float> expected = {
        // Channel 1
        2.29426f, 3.89173f, 4.17634f, 3.25501f, 2.07618f, 5.41483f, 7.09971f,
        6.39811f, 5.71432f, 3.10928f, 5.12973f, 6.29638f, 5.26962f, 5.21997f,
        3.05852f, 6.17517f, 7.19311f, 6.69771f, 6.2142f, 4.03242f, 3.3792f,
        4.36444f, 4.396f, 4.69905f, 3.62061f,
        // Channel 2
        2.87914f, 3.71743f, 3.51854f, 2.98413f, 1.46579f, 4.94951f, 6.18983f,
        4.98187f, 4.38372f, 3.35386f, 5.0364f, 5.3756f, 4.05993f, 4.89299f,
        2.78625f, 5.33763f, 5.80899f, 5.89785f, 5.51095f, 3.74287f, 2.64053f,
        4.05895f, 3.96482f, 4.30177f, 1.94269f
    };
    for (int i = 0; i < output.size(); i++) {
        EXPECT_NEAR(output[i], expected[i], 0.0001f);
    }

    commonTestTeardown(d_input);
}

TEST_F(Conv2dTest, StridedPaddedConvolution) {
    int                         inputSize     = 5;
    int                         inputChannels = 2;
    int                         kernelSize    = 3;
    int                         stride        = 2;
    int                         numFilters    = 2;
    CUDANet::Layers::Padding    padding       = CUDANet::Layers::Padding::SAME;
    CUDANet::Layers::Activation activation = CUDANet::Layers::Activation::RELU;

    // clang-format off
    std::vector<float> input = {
        // Channel 1
        0.946f, 0.879f, 0.382f, 0.542f, 0.453f,
        0.128f, 0.860f, 0.778f, 0.049f, 0.974f,
        0.400f, 0.874f, 0.161f, 0.271f, 0.580f,
        0.373f, 0.078f, 0.366f, 0.396f, 0.181f,
        0.246f, 0.112f, 0.179f, 0.979f, 0.026f,
        // Channel 2
        0.598f, 0.458f, 0.776f, 0.213f, 0.199f,
        0.853f, 0.170f, 0.609f, 0.269f, 0.777f,
        0.776f, 0.694f, 0.430f, 0.238f, 0.968f,
        0.473f, 0.303f, 0.084f, 0.785f, 0.444f,
        0.464f, 0.413f, 0.779f, 0.298f, 0.783f
    };
    std::vector<float> kernels = {
        // Filter 1, Channel 1
        0.744f, 0.745f, 0.641f,
        0.164f, 0.157f, 0.127f,
        0.732f, 0.761f, 0.601f,
        // Filter 1, Channel 2
        0.475f, 0.335f, 0.499f,
        0.833f, 0.793f, 0.176f,
        0.822f, 0.163f, 0.175f,
        // Filter 2, Channel 1
        0.918f, 0.340f, 0.497f,
        0.233f, 0.218f, 0.847f,
        0.931f, 0.926f, 0.199f,
        // Filter 2, Channel 2
        0.510f, 0.432f, 0.567f,
        0.236f, 0.397f, 0.739f,
        0.939f, 0.891f, 0.006f
    };
    // clang-format on

    float* d_input;
    float* d_output;

    CUDANet::Layers::Conv2d conv2d = commonTestSetup(
        inputSize, inputChannels, kernelSize, stride, padding, numFilters,
        activation, input, kernels.data(), d_input
    );

    EXPECT_EQ(inputSize, conv2d.getOutputSize());

    d_output = conv2d.forward(d_input);

    std::vector<float> output(
        conv2d.getOutputSize() * conv2d.getOutputSize() * numFilters
    );
    hipMemcpy(
        output.data(), d_output,
        sizeof(float) * conv2d.getOutputSize() * conv2d.getOutputSize() *
            numFilters,
        hipMemcpyDeviceToHost
    );

    // Generated by tools/generate_conv2d_test.py
    std::vector<float> expected = {
        // Channel 1
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 1.59803f, 2.84444f, 1.6201f, 0.0f,
        0.0f, 2.38937f, 3.80762f, 3.39679f, 0.0f, 0.0f, 1.13102f, 2.33335f,
        1.98488f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        // Channel 2
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 2.57732f, 3.55543f, 2.24675f, 0.0f,
        0.0f, 3.36842f, 3.41373f, 3.14804f, 0.0f, 0.0f, 1.17963f, 2.55005f,
        1.63218f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f
    };

    for (int i = 0; i < output.size(); i++) {
        EXPECT_NEAR(output[i], expected[i], 0.0001f);
    }

    commonTestTeardown(d_input);
}
