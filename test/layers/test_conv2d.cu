#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "conv2d.cuh"

TEST(Conv2dTest, SimpleExample) {

    int inputSize = 4;
    int inputChannels = 1;
    int kernelSize = 2;
    int stride = 1;
    std::string padding = "VALID";
    int numFilters = 1;
    Activation activation = LINEAR;

    Layers::Conv2d conv2d(
        inputSize,
        inputChannels,
        kernelSize,
        stride,
        padding,
        numFilters,
        activation
    );

    int outputSize = (inputSize - kernelSize) / stride + 1;
    EXPECT_EQ(outputSize, conv2d.outputSize);

    std::vector<float> input = {
        1.0f, 2.0f, 3.0f, 4.0f,
        5.0f, 6.0f, 7.0f, 8.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        13.0f, 14.0f, 15.0f, 16.0f
    };
    std::vector<float> kernels = {
        1.0f, 2.0f, 3.0f, 4.0f,
    };

    conv2d.setKernels(kernels);

    
    std::vector<float> output(outputSize * outputSize * numFilters);

    conv2d.host_conv(input.data(), output.data());

    std::vector<float> expected = {
        44.0f, 54.0f, 64.0f,
        84.0f, 94.0f, 104.0f,
        124.0f, 134.0f, 144.0f
    };  

    for (int i = 0; i < output.size(); ++i) {
        EXPECT_FLOAT_EQ(expected[i], output[i]);
    }

}
