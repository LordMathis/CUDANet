#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "conv2d.cuh"

class Conv2dTest : public ::testing::Test {
  protected:
    shape2d                           inputSize;
    int                             inputChannels;
    shape2d                           kernelSize;
    shape2d                           stride;
    int                             numFilters;
    shape2d                           paddingSize;
    CUDANet::Layers::ActivationType activationType;
    std::vector<float>              input;
    std::vector<float>              kernels;
    std::vector<float>              expected;

    float                   *d_input;
    float                   *d_output;
    CUDANet::Layers::Conv2d *conv2dLayer;

    virtual void SetUp() override {
        d_input     = nullptr;
        d_output    = nullptr;
        conv2dLayer = nullptr;
    }

    virtual void TearDown() override {
        if (d_input) {
            hipFree(d_input);
        }
        delete conv2dLayer;
    }

    void runTest() {
        hipError_t cudaStatus;

        conv2dLayer = new CUDANet::Layers::Conv2d(
            inputSize, inputChannels, kernelSize, stride, numFilters,
            paddingSize, activationType
        );

        conv2dLayer->setWeights(kernels.data());

        cudaStatus =
            hipMalloc((void **)&d_input, sizeof(float) * input.size());
        EXPECT_EQ(cudaStatus, hipSuccess);

        cudaStatus = hipMemcpy(
            d_input, input.data(), sizeof(float) * input.size(),
            hipMemcpyHostToDevice
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        d_output = conv2dLayer->forward(d_input);

        int outputHeight =
            (inputSize.first - kernelSize.first + 2 * paddingSize.first) /
                stride.first +
            1;
        int outputWidth =
            (inputSize.second - kernelSize.second + 2 * paddingSize.second) /
                stride.second +
            1;
        int outputSize = outputHeight * outputWidth * numFilters;
        EXPECT_EQ(outputSize, conv2dLayer->getOutputSize());

        std::vector<float> output(outputSize);
        cudaStatus = hipMemcpy(
            output.data(), d_output, sizeof(float) * output.size(),
            hipMemcpyDeviceToHost
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        for (int i = 0; i < output.size(); ++i) {
            EXPECT_NEAR(expected[i], output[i], 1e-5f);
        }
    }
};

TEST_F(Conv2dTest, SimpleTest) {
    inputSize      = {4, 4};
    inputChannels  = 1;
    kernelSize     = {2, 2};
    stride         = {1, 1};
    numFilters     = 1;
    paddingSize    = {0, 0};
    activationType = CUDANet::Layers::ActivationType::NONE;

    input = {
        // clang-format off
        1.0f, 2.0f, 3.0f, 4.0f,
        5.0f, 6.0f, 7.0f, 8.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        13.0f, 14.0f, 15.0f, 16.0f
        // clang-format on
    };
    kernels = {
        // clang-format off
        1.0f,2.0f,
        3.0f, 4.0f
        // clang-format on
    };
    expected = {44.0f,  54.0f,  64.0f,  84.0f, 94.0f,
                104.0f, 124.0f, 134.0f, 144.0f};

    runTest();
}

TEST_F(Conv2dTest, PaddedTest) {
    inputSize     = {5, 5};
    inputChannels = 3;
    kernelSize    = {3, 3};
    stride        = {1, 1};
    numFilters    = 2;

    int paddingFirst =
        CUDANET_SAME_PADDING(inputSize.first, kernelSize.first, stride.first);
    int paddingSecond = CUDANET_SAME_PADDING(
        inputSize.second, kernelSize.second, stride.second
    );
    paddingSize = {paddingFirst, paddingSecond};

    activationType = CUDANet::Layers::ActivationType::NONE;

    // clang-format off
    input = {
        // Channel 1
        0.823f, 0.217f, 0.435f, 0.981f, 0.742f,
        0.109f, 0.518f, 0.374f, 0.681f, 0.147f,
        0.956f, 0.729f, 0.654f, 0.087f, 0.392f,
        0.784f, 0.921f, 0.543f, 0.231f, 0.816f,
        0.472f, 0.614f, 0.102f, 0.987f, 0.398f,
        // Channel 2
        0.051f, 0.756f, 0.841f, 0.293f, 0.128f,
        0.417f, 0.632f, 0.095f, 0.184f, 0.529f,
        0.871f, 0.958f, 0.213f, 0.347f, 0.725f,
        0.461f, 0.012f, 0.278f, 0.195f, 0.649f,
        0.853f, 0.707f, 0.988f, 0.988f, 0.322f,
        // Channel 3
        0.345f, 0.123f, 0.789f, 0.123f, 0.456f,
        0.456f, 0.789f, 0.123f, 0.345f, 0.123f,
        0.789f, 0.123f, 0.345f, 0.123f, 0.456f,
        0.123f, 0.345f, 0.123f, 0.789f, 0.123f,
        0.345f, 0.123f, 0.789f, 0.123f, 0.456f
    };

    kernels = {
        // Filter 1, Channel 1
        0.128f, 0.754f, 0.987f,
        0.321f, 0.412f, 0.635f,
        0.298f, 0.017f, 0.845f,
        // Filter 1, Channel 2
        0.514f, 0.729f, 0.952f,
        0.684f, 0.378f, 0.159f,
        0.823f, 0.547f, 0.216f,
        // Filter 1, Channel 3
        0.983f, 0.231f, 0.456f,
        0.178f,  0.654f, 0.821f,
        0.345f, 0.987f, 0.123f,
        // Filter 2, Channel 1
        0.789f, 0.543f, 0.210f,
        0.012f, 0.371f, 0.638f,
        0.456f, 0.198f, 0.907f,
        // Filter 2, Channel 2
        0.101f, 0.432f, 0.759f,
        0.234f, 0.567f, 0.890f,
        0.543f, 0.876f, 0.219f,
        // Filter 2, Channel 3
        0.345f, 0.678f, 0.011f,
        0.678f, 0.011f, 0.345f,
        0.011f, 0.345f, 0.678f
    };
    // clang-format on

    // Generated by tools/generate_conv2d_test.py
    expected = {
        // Channel 1
        2.29426f, 3.89173f, 4.17634f, 3.25501f, 2.07618f, 5.41483f, 7.09971f,
        6.39811f, 5.71432f, 3.10928f, 5.12973f, 6.29638f, 5.26962f, 5.21997f,
        3.05852f, 6.17517f, 7.19311f, 6.69771f, 6.2142f, 4.03242f, 3.3792f,
        4.36444f, 4.396f, 4.69905f, 3.62061f,
        // Channel 2
        2.87914f, 3.71743f, 3.51854f, 2.98413f, 1.46579f, 4.94951f, 6.18983f,
        4.98187f, 4.38372f, 3.35386f, 5.0364f, 5.3756f, 4.05993f, 4.89299f,
        2.78625f, 5.33763f, 5.80899f, 5.89785f, 5.51095f, 3.74287f, 2.64053f,
        4.05895f, 3.96482f, 4.30177f, 1.94269f
    };

    runTest();
}

TEST_F(Conv2dTest, StridedPaddedTest) {
    inputSize     = {5, 5};
    inputChannels = 2;
    kernelSize    = {3, 3};
    stride        = {2, 2};
    numFilters    = 2;

    int paddingFirst =
        CUDANET_SAME_PADDING(inputSize.first, kernelSize.second, stride.first);
    int paddingSecond = CUDANET_SAME_PADDING(
        inputSize.second, kernelSize.second, stride.second
    );
    paddingSize = {paddingFirst, paddingSecond};

    activationType = CUDANet::Layers::ActivationType::RELU;

    // clang-format off
    input = {
        // Channel 1
        0.946f, 0.879f, 0.382f, 0.542f, 0.453f,
        0.128f, 0.860f, 0.778f, 0.049f, 0.974f,
        0.400f, 0.874f, 0.161f, 0.271f, 0.580f,
        0.373f, 0.078f, 0.366f, 0.396f, 0.181f,
        0.246f, 0.112f, 0.179f, 0.979f, 0.026f,
        // Channel 2
        0.598f, 0.458f, 0.776f, 0.213f, 0.199f,
        0.853f, 0.170f, 0.609f, 0.269f, 0.777f,
        0.776f, 0.694f, 0.430f, 0.238f, 0.968f,
        0.473f, 0.303f, 0.084f, 0.785f, 0.444f,
        0.464f, 0.413f, 0.779f, 0.298f, 0.783f
    };
    kernels = {
        // Filter 1, Channel 1
        0.744f, 0.745f, 0.641f,
        0.164f, 0.157f, 0.127f,
        0.732f, 0.761f, 0.601f,
        // Filter 1, Channel 2
        0.475f, 0.335f, 0.499f,
        0.833f, 0.793f, 0.176f,
        0.822f, 0.163f, 0.175f,
        // Filter 2, Channel 1
        0.918f, 0.340f, 0.497f,
        0.233f, 0.218f, 0.847f,
        0.931f, 0.926f, 0.199f,
        // Filter 2, Channel 2
        0.510f, 0.432f, 0.567f,
        0.236f, 0.397f, 0.739f,
        0.939f, 0.891f, 0.006f
    };
    // clang-format on

    expected = {// Channel 1
                0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 1.59803f, 2.84444f, 1.6201f,
                0.0f, 0.0f, 2.38937f, 3.80762f, 3.39679f, 0.0f, 0.0f, 1.13102f,
                2.33335f, 1.98488f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                // Channel 2
                0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 2.57732f, 3.55543f,
                2.24675f, 0.0f, 0.0f, 3.36842f, 3.41373f, 3.14804f, 0.0f, 0.0f,
                1.17963f, 2.55005f, 1.63218f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f
    };

    runTest();
}

TEST_F(Conv2dTest, NonSquareInputTest) {
    inputSize      = {4, 6};  // Non-square input
    inputChannels  = 1;
    kernelSize     = {2, 2};
    stride         = {1, 1};
    numFilters     = 1;
    paddingSize    = {0, 0};
    activationType = CUDANet::Layers::ActivationType::NONE;

    input = {
        // clang-format off
        0.946f, 0.879f, 0.382f, 0.542f, 0.453f, 0.128f,
        0.128f, 0.860f, 0.778f, 0.049f, 0.974f, 0.400f,
        0.400f, 0.874f, 0.161f, 0.271f, 0.580f, 0.373f,
        0.078f, 0.366f, 0.396f, 0.181f, 0.246f, 0.112f
        // clang-format on
    };
    kernels  = {0.744f, 0.745f, 0.164f, 0.157f};
    expected = {1.51469f, 1.20175f, 0.82328f, 0.90169f, 0.65493f,
                0.93875f, 1.38806f, 0.68429f, 0.89759f, 1.17634f,
                1.01898f, 0.8924f,  0.41504f, 0.70203f, 0.76733f};

    runTest();
}

TEST_F(Conv2dTest, NonSquareKernelTest) {
    inputSize      = {4, 4};
    inputChannels  = 1;
    kernelSize     = {1, 3};  // Non-square kernel
    stride         = {1, 1};
    numFilters     = 1;
    paddingSize    = {0, 0};
    activationType = CUDANet::Layers::ActivationType::NONE;

    input = {
        // clang-format off
        0.946f, 0.879f, 0.382f, 0.542f,
        0.128f, 0.860f, 0.778f, 0.049f,
        0.400f, 0.874f, 0.161f, 0.271f,
        0.078f, 0.366f, 0.396f, 0.181f
        // clang-format on
    };
    kernels  = {0.744f, 0.745f, 0.164f};
    expected = {1.42133f, 1.02745f, 0.86352f, 1.22749f,
                0.97513f, 0.81465f, 0.39565f, 0.59701f};

    runTest();
}

TEST_F(Conv2dTest, NonSquareStrideTest) {
    inputSize      = {4, 4};
    inputChannels  = 1;
    kernelSize     = {2, 2};
    stride         = {1, 2};  // Non-square stride
    numFilters     = 1;
    paddingSize    = {0, 0};
    activationType = CUDANet::Layers::ActivationType::NONE;

    input = {
        // clang-format off
        0.946f, 0.879f, 0.382f, 0.542f,
        0.128f, 0.860f, 0.778f, 0.049f,
        0.400f, 0.874f, 0.161f, 0.271f,
        0.078f, 0.366f, 0.396f, 0.181f
        // clang-format on
    };
    kernels  = {0.144f, 0.745f, 0.964f, 0.164f};
    expected = {1.05551f, 1.21683f, 1.18807f, 0.34818f, 0.84395f, 0.63651f};

    runTest();
}

TEST_F(Conv2dTest, NonSquarePaddingTest) {
    inputSize      = {4, 4};
    inputChannels  = 1;
    kernelSize     = {2, 2};
    stride         = {1, 1};
    numFilters     = 1;
    paddingSize    = {1, 2};  // Non-square padding
    activationType = CUDANet::Layers::ActivationType::NONE;

    input = {
        // clang-format off
        0.946f, 0.879f, 0.382f, 0.542f,
        0.128f, 0.860f, 0.778f, 0.049f,
        0.400f, 0.874f, 0.161f, 0.271f,
        0.078f, 0.366f, 0.396f, 0.181f
        // clang-format on
    };
    kernels  = {0.144f, 0.745f, 0.964f, 0.164f};
    expected = {0.0f, 0.15514f, 1.0561f,  0.91f,    0.45714f, 0.52249f, 0.0f,
                0.0f, 0.72576f, 1.05551f, 1.3678f,  1.21683f, 0.12528f, 0.0f,
                0.0f, 0.16096f, 1.18807f, 1.57239f, 0.34818f, 0.2683f,  0.0f,
                0.0f, 0.31079f, 0.84395f, 0.66357f, 0.63651f, 0.21351f, 0.0f,
                0.0f, 0.05811f, 0.2839f,  0.34772f, 0.19187f, 0.02606f, 0.0f};
}