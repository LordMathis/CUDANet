#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>

#include <iostream>

#include "conv2d.cuh"

class Conv2dTest : public::testing::Test {
    protected:
        hipError_t    cudaStatus;
};


TEST_F(Conv2dTest, SimpleExample) {

    int inputSize = 4;
    int inputChannels = 1;
    int kernelSize = 2;
    int stride = 1;
    std::string padding = "VALID";
    int numFilters = 1;
    Activation activation = LINEAR;

    Layers::Conv2d conv2d(
        inputSize,
        inputChannels,
        kernelSize,
        stride,
        padding,
        numFilters,
        activation
    );

    int outputSize = (inputSize - kernelSize) / stride + 1;
    EXPECT_EQ(outputSize, conv2d.outputSize);

    std::vector<float> input = {
        1.0f, 2.0f, 3.0f, 4.0f,
        5.0f, 6.0f, 7.0f, 8.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        13.0f, 14.0f, 15.0f, 16.0f
    };
    std::vector<float> kernels = {
        1.0f, 2.0f, 3.0f, 4.0f,
    };

    float* d_input;
    float* d_output;

    conv2d.setKernels(kernels);

            // Allocate device memory
    cudaStatus = hipMalloc((void**)&d_input, sizeof(float) * inputSize * inputSize * inputChannels);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_output, sizeof(float) * outputSize * outputSize * numFilters);
    EXPECT_EQ(cudaStatus, hipSuccess);

    // // Copy input to device
    cudaStatus = hipMemcpy(
        d_input, input.data(), sizeof(float) * input.size(), hipMemcpyHostToDevice
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    conv2d.forward(d_input, d_output);

    std::vector<float> expected = {
        44.0f, 54.0f, 64.0f,
        84.0f, 94.0f, 104.0f,
        124.0f, 134.0f, 144.0f
    };
    std::vector<float> output(outputSize * outputSize * numFilters);

    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * output.size(),
        hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < output.size(); ++i) {
        EXPECT_FLOAT_EQ(expected[i], output[i]);
    }

}
