#include "concat.cuh"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>

TEST(ConcatLayerTest, Init) {
    std::vector<float> inputA = {0.573f, 0.619f, 0.732f, 0.055f, 0.243f};
    std::vector<float> inputB = {0.123f, 0.321f, 0.456f, 0.789f, 0.654f, 0.123f};

    CUDANet::Layers::Concat concat(5, 6);   

    float* d_inputA;
    float* d_inputB;
    hipMalloc((void**)&d_inputA, sizeof(float) * 5);
    hipMalloc((void**)&d_inputB, sizeof(float) * 6);
    hipMemcpy(
        d_inputA, inputA.data(), sizeof(float) * 5, hipMemcpyHostToDevice
    );
    hipMemcpy(
        d_inputB, inputB.data(), sizeof(float) * 6, hipMemcpyHostToDevice
    );

    float* d_output = concat.forward(d_inputA, d_inputB);

    std::vector<float> output(11);
    hipMemcpy(
        output.data(), d_output, sizeof(float) * 11, hipMemcpyDeviceToHost
    );

    for (int i = 0; i < 5; ++i) {
        EXPECT_EQ(output[i], inputA[i]);
    }
    for (int i = 0; i < 6; ++i) {
        EXPECT_EQ(output[i + 5], inputB[i]);
    }
    hipFree(d_output);

    hipDeviceReset();
}