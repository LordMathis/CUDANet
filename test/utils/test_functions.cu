#include "gtest/gtest.h"
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include "functions.cuh"
#include "test_cublas_fixture.cuh"

class FunctionsTest : public CublasTestFixture {
protected:
    hipError_t cudaStatus;
    hipblasStatus_t hipblasStatus_t;
};

TEST_F(FunctionsTest, sigmoid) {

}