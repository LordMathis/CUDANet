#include "gtest/gtest.h"
#include "hipblas.h"
#include "test_cublas_fixture.cuh"

hipblasHandle_t CublasTestFixture::cublasHandle;

void CublasTestFixture::SetUpTestSuite() {
    hipblasCreate(&cublasHandle);
}

void CublasTestFixture::TearDownTestSuite() {
    hipblasDestroy(cublasHandle);
}
