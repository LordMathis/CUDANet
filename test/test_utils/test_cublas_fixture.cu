#include <hipblas.h>
#include <gtest/gtest.h>

#include "test_cublas_fixture.cuh"

hipblasHandle_t CublasTestFixture::cublasHandle;

void CublasTestFixture::SetUpTestSuite() {
    hipblasCreate(&cublasHandle);
}

void CublasTestFixture::TearDownTestSuite() {
    hipblasDestroy(cublasHandle);
}
