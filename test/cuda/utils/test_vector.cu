#include <gtest/gtest.h>

#include "vector.cuh"

TEST(VectorTest, TestVectorMean) {

    hipError_t cudaStatus;
    float length = 10;

    std::vector<float> input = {0.44371f, 0.20253f, 0.73232f, 0.40378f, 0.93348f, 0.72756f, 0.63388f, 0.5251f, 0.23973f, 0.52233f};

    float* d_vec = nullptr;
    cudaStatus = hipMalloc((void **)&d_vec, sizeof(float) * length);
    EXPECT_EQ(cudaStatus, hipSuccess);

    float* d_mean = nullptr;
    cudaStatus = hipMalloc((void **)&d_mean, sizeof(float) * length);
    EXPECT_EQ(cudaStatus, hipSuccess);
    
    float* d_length = nullptr;
    cudaStatus = hipMalloc((void **)&d_length, sizeof(float));
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMemcpy(d_vec, input.data(), sizeof(float) * length, hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMemcpy(d_length, &length, sizeof(float), hipMemcpyHostToDevice);
    EXPECT_EQ(cudaStatus, hipSuccess);

    CUDANet::Utils::mean(d_vec, d_mean, d_length, length);

    std::vector<float> mean(length);
    cudaStatus = hipMemcpy(mean.data(), d_mean, sizeof(float) * length, hipMemcpyDeviceToHost);
    EXPECT_EQ(cudaStatus, hipSuccess);

    float expected_mean = 0.5364f;
    EXPECT_NEAR(mean[0], expected_mean, 1e-4);

}