#include "concat.hpp"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>

TEST(ConcatLayerTest, Init) {

    hipError_t cudaStatus;

    std::vector<float> inputA = {0.573f, 0.619f, 0.732f, 0.055f, 0.243f};
    std::vector<float> inputB = {0.123f, 0.321f, 0.456f, 0.789f, 0.654f, 0.123f};

    CUDANet::Layers::Concat concat(5, 6);   

    float* d_inputA;
    float* d_inputB;

    cudaStatus = hipMalloc((void**)&d_inputA, sizeof(float) * 5);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMalloc((void**)&d_inputB, sizeof(float) * 6);
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMemcpy(
        d_inputA, inputA.data(), sizeof(float) * 5, hipMemcpyHostToDevice
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    cudaStatus = hipMemcpy(
        d_inputB, inputB.data(), sizeof(float) * 6, hipMemcpyHostToDevice
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    float* d_output = concat.forward(d_inputA, d_inputB);

    std::vector<float> output(11);
    cudaStatus = hipMemcpy(
        output.data(), d_output, sizeof(float) * 11, hipMemcpyDeviceToHost
    );
    EXPECT_EQ(cudaStatus, hipSuccess);

    for (int i = 0; i < 5; ++i) {
        EXPECT_EQ(output[i], inputA[i]);
    }

    for (int i = 0; i < 6; ++i) {
        EXPECT_EQ(output[i + 5], inputB[i]);
    }

    cudaStatus = hipFree(d_inputA);
    EXPECT_EQ(cudaStatus, hipSuccess);
    cudaStatus = hipFree(d_inputB);
    EXPECT_EQ(cudaStatus, hipSuccess);
}