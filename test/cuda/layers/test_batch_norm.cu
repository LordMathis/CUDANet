#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <vector>

#include "activation.hpp"
#include "batch_norm.hpp"

class BatchNormLayerTest : public ::testing::Test {
  protected:
    shape2d            inputSize;
    int                nChannels;
    std::vector<float> weights;
    std::vector<float> biases;

    std::vector<float> runningMean;
    std::vector<float> runningVar;

    std::vector<float> input;
    std::vector<float> expected;

    float                        *d_input;
    float                        *d_output;
    CUDANet::Layers::BatchNorm2d *batchNorm;

    virtual void SetUp() override {
        d_input   = nullptr;
        d_output  = nullptr;
        batchNorm = nullptr;
    }

    virtual void TearDown() override {
        if (d_input) {
            hipFree(d_input);
        }
    }

    void runTest() {
        hipError_t cudaStatus;

        batchNorm = new CUDANet::Layers::BatchNorm2d(
            inputSize, nChannels, 1e-5f, CUDANet::Layers::ActivationType::NONE
        );

        batchNorm->setWeights(weights.data());
        batchNorm->setBiases(biases.data());

        batchNorm->setRunningMean(runningMean.data());
        batchNorm->setRunningVar(runningVar.data());

        cudaStatus = hipGetLastError();
        EXPECT_EQ(cudaStatus, hipSuccess);

        cudaStatus =
            hipMalloc((void **)&d_input, sizeof(float) * input.size());
        EXPECT_EQ(cudaStatus, hipSuccess);

        cudaStatus = hipMemcpy(
            d_input, input.data(), sizeof(float) * input.size(),
            hipMemcpyHostToDevice
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        d_output = batchNorm->forward(d_input);

        std::vector<float> output(input.size());
        cudaStatus = hipMemcpy(
            output.data(), d_output, sizeof(float) * output.size(),
            hipMemcpyDeviceToHost
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        for (int i = 0; i < output.size(); ++i) {
            EXPECT_NEAR(output[i], expected[i], 1e-5);
        }

        delete batchNorm;
    }
};

TEST_F(BatchNormLayerTest, BatchNormSmallForwardTest) {
    inputSize = {4, 4};
    nChannels = 2;

    weights = {0.63508f, 0.64903f};
    biases  = {0.25079f, 0.66841f};

    runningMean = {0.5f, 0.5f};
    runningVar  = {1.0f, 1.0f};

    // clang-format off
    input = {
        // Channel 0
        0.38899f, 0.80478f, 0.48836f, 0.97381f,
        0.57508f, 0.60835f, 0.65467f, 0.00168f,
        0.65869f, 0.74235f, 0.17928f, 0.70349f,
        0.15524f, 0.38664f, 0.23411f, 0.7137f,
        // Channel 1
        0.32473f, 0.15698f, 0.314f, 0.60888f,
        0.80268f, 0.99766f, 0.93694f, 0.89237f,
        0.13449f, 0.27367f, 0.53036f, 0.18962f,
        0.57672f, 0.48364f, 0.10863f, 0.0571f
    };
    // clang-format on

    expected = {0.18029f, 0.44435f,  0.2434f,  0.5517f,  0.29847f, 0.3196f,
                0.34902f, -0.06568f, 0.35157f, 0.4047f,  0.04711f, 0.38002f,
                0.03184f, 0.1788f,   0.08193f, 0.38651f, 0.55466f, 0.44578f,
                0.54769f, 0.73908f,  0.86486f, 0.9914f,  0.952f,   0.92307f,
                0.43118f, 0.52152f,  0.68811f, 0.46697f, 0.7182f,  0.65779f,
                0.4144f,  0.38096f};

    runTest();
}

TEST_F(BatchNormLayerTest, BatchNormNonSquareInputTest) {
    inputSize = {4, 6};  // Non-square input
    nChannels = 2;
    weights   = {0.63508f, 0.64903f};
    biases    = {0.25079f, 0.66841f};

    runningMean = {0.5f, 0.5f};
    runningVar  = {1.0f, 1.0f};

    input = {// Channel 0
             0.38899f, 0.80478f, 0.48836f, 0.97381f, 0.21567f, 0.92312f,
             0.57508f, 0.60835f, 0.65467f, 0.00168f, 0.31567f, 0.71345f,
             0.65869f, 0.74235f, 0.17928f, 0.70349f, 0.12856f, 0.95645f,
             0.15524f, 0.38664f, 0.23411f, 0.7137f, 0.26789f, 0.83412f,
             // Channel 1
             0.32473f, 0.15698f, 0.314f, 0.60888f, 0.23145f, 0.78945f, 0.80268f,
             0.99766f, 0.93694f, 0.89237f, 0.61234f, 0.92314f, 0.13449f,
             0.27367f, 0.53036f, 0.18962f, 0.45623f, 0.14523f, 0.57672f,
             0.48364f, 0.10863f, 0.0571f, 0.78934f, 0.67545f
    };

    expected = {0.18029f, 0.44435f, 0.2434f,  0.5517f,   0.07022f, 0.5195f,
                0.29847f, 0.3196f,  0.34902f, -0.06568f, 0.13373f, 0.38635f,
                0.35157f, 0.4047f,  0.04711f, 0.38002f,  0.0149f,  0.54067f,
                0.03184f, 0.1788f,  0.08193f, 0.38651f,  0.10338f, 0.46298f,
                0.55466f, 0.44578f, 0.54769f, 0.73908f,  0.49411f, 0.85627f,
                0.86486f, 0.9914f,  0.952f,   0.92307f,  0.74132f, 0.94304f,
                0.43118f, 0.52152f, 0.68811f, 0.46697f,  0.64f,    0.43815f,
                0.7182f,  0.65779f, 0.4144f,  0.38096f,  0.8562f,  0.78228f};

    runTest();
}