#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <vector>

#include "max_pooling.hpp"

class MaxPoolingLayerTest : public ::testing::Test {
  protected:
    shape2d              inputSize;
    int                nChannels;
    shape2d              poolingSize;
    shape2d              stride;
    shape2d              padding;
    std::vector<float> input;
    std::vector<float> expected;

    float                         *d_input;
    float                         *d_output;
    CUDANet::Layers::MaxPooling2d *maxPoolingLayer;

    virtual void SetUp() override {
        d_input         = nullptr;
        d_output        = nullptr;
        maxPoolingLayer = nullptr;
    }

    virtual void TearDown() override {
        if (d_input) {
            hipFree(d_input);
        }
        delete maxPoolingLayer;
    }

    void runTest() {
        hipError_t cudaStatus;

        maxPoolingLayer = new CUDANet::Layers::MaxPooling2d(
            inputSize, nChannels, poolingSize, stride, padding,
            CUDANet::Layers::ActivationType::NONE
        );

        cudaStatus =
            hipMalloc((void **)&d_input, sizeof(float) * input.size());
        EXPECT_EQ(cudaStatus, hipSuccess);

        cudaStatus = hipMemcpy(
            d_input, input.data(), sizeof(float) * input.size(),
            hipMemcpyHostToDevice
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        d_output = maxPoolingLayer->forward(d_input);

        int outputSize = maxPoolingLayer->getOutputSize();

        std::vector<float> output(outputSize);
        cudaStatus = hipMemcpy(
            output.data(), d_output, sizeof(float) * output.size(),
            hipMemcpyDeviceToHost
        );
        EXPECT_EQ(cudaStatus, hipSuccess);

        for (int i = 0; i < output.size(); ++i) {
            EXPECT_FLOAT_EQ(expected[i], output[i]);
        }
    }
};

TEST_F(MaxPoolingLayerTest, MaxPoolForwardTest) {
    inputSize   = {4, 4};
    nChannels   = 2;
    poolingSize = {2, 2};
    stride      = {2, 2};
    padding     = {0, 0};

    input = {
        // clang-format off
        // Channel 0
        0.573f, 0.619f, 0.732f, 0.055f,
        0.243f, 0.316f, 0.573f, 0.619f,
        0.712f, 0.055f, 0.243f, 0.316f,
        0.573f, 0.619f, 0.742f, 0.055f,
        // Channel 1
        0.473f, 0.919f, 0.107f, 0.073f,
        0.073f, 0.362f, 0.973f, 0.059f,
        0.473f, 0.455f, 0.283f, 0.416f,
        0.532f, 0.819f, 0.732f, 0.850f
        // clang-format on
    };

    expected = {0.619f, 0.732f, 0.712f, 0.742f, 0.919f, 0.973f, 0.819f, 0.85f};

    runTest();
}

TEST_F(MaxPoolingLayerTest, MaxPoolForwardNonSquareInputTest) {
    inputSize   = {4, 6};  // Non-square input
    nChannels   = 2;
    poolingSize = {2, 2};
    stride      = {2, 2};
    padding     = {0, 0};

    input = {// Channel 0
             0.573f, 0.619f, 0.732f, 0.055f, 0.123f, 0.234f, 0.243f, 0.316f,
             0.573f, 0.619f, 0.456f, 0.789f, 0.712f, 0.055f, 0.243f, 0.316f,
             0.654f, 0.987f, 0.573f, 0.619f, 0.742f, 0.055f, 0.321f, 0.654f,
             // Channel 1
             0.473f, 0.919f, 0.107f, 0.073f, 0.321f, 0.654f, 0.073f, 0.362f,
             0.973f, 0.059f, 0.654f, 0.987f, 0.473f, 0.455f, 0.283f, 0.416f,
             0.789f, 0.123f, 0.532f, 0.819f, 0.732f, 0.850f, 0.987f, 0.321f
    };

    expected = {0.619f, 0.732f, 0.789f, 0.712f, 0.742f, 0.987f, 0.919f, 0.973f, 0.987f, 0.819f, 0.85f, 0.987f};

    runTest();
}

TEST_F(MaxPoolingLayerTest, MaxPoolForwardNonSquarePoolSizeTest) {
    inputSize   = {4, 4};
    nChannels   = 2;
    poolingSize = {2, 3};  // Non-square pooling size
    stride      = {2, 2};
    padding     = {0, 0};

    input = {
        // clang-format off
        // Channel 0
        0.573f, 0.619f, 0.732f, 0.055f,
        0.243f, 0.316f, 0.573f, 0.619f,
        0.712f, 0.055f, 0.243f, 0.316f,
        0.573f, 0.619f, 0.742f, 0.055f,
        // Channel 1
        0.473f, 0.919f, 0.107f, 0.073f,
        0.073f, 0.362f, 0.973f, 0.059f,
        0.473f, 0.455f, 0.283f, 0.416f,
        0.532f, 0.819f, 0.732f, 0.850f
        // clang-format on
    };

    expected = {0.732f, 0.742f, 0.973f, 0.819f};

    runTest();

}

TEST_F(MaxPoolingLayerTest, MaxPoolForwardNonSquareStrideTest) {
    inputSize   = {4, 4};
    nChannels   = 2;
    poolingSize = {2, 2};
    stride      = {1, 2};  // Non-square stride
    padding     = {0, 0};

    input = {
        // clang-format off
        // Channel 0
        0.573f, 0.619f, 0.732f, 0.055f,
        0.243f, 0.316f, 0.573f, 0.619f,
        0.712f, 0.055f, 0.243f, 0.316f,
        0.573f, 0.619f, 0.742f, 0.055f,
        // Channel 1
        0.473f, 0.919f, 0.107f, 0.073f,
        0.073f, 0.362f, 0.973f, 0.059f,
        0.473f, 0.455f, 0.283f, 0.416f,
        0.532f, 0.819f, 0.732f, 0.850f
        // clang-format on
    };

    expected = {0.619f, 0.732f, 0.712f, 0.619f, 0.712f, 0.742f, 0.919f, 0.973f, 0.473f, 0.973f, 0.819f, 0.85f};

    runTest();

}

TEST_F(MaxPoolingLayerTest, MaxPoolForwardNonSquarePaddingTest) {
    inputSize   = {4, 4};
    nChannels   = 2;
    poolingSize = {2, 2};
    stride      = {2, 2};  // Non-square stride
    padding     = {0, 1};

    input = {
        // clang-format off
        // Channel 0
        0.573f, 0.619f, 0.732f, 0.055f,
        0.243f, 0.316f, 0.573f, 0.619f,
        0.712f, 0.055f, 0.243f, 0.316f,
        0.573f, 0.619f, 0.742f, 0.055f,
        // Channel 1
        0.473f, 0.919f, 0.107f, 0.073f,
        0.073f, 0.362f, 0.973f, 0.059f,
        0.473f, 0.455f, 0.283f, 0.416f,
        0.532f, 0.819f, 0.732f, 0.850f
        // clang-format on
    };

    expected = {0.573f, 0.732f, 0.619f, 0.712f, 0.742f, 0.316f, 0.473f, 0.973f, 0.073f, 0.532f, 0.819f, 0.85f};

    runTest();

}