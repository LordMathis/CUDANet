#include <string>

#include "activations.cuh"
#include "conv2d.cuh"
#include "cuda_helper.cuh"
#include "padding.cuh"

Layers::Conv2d::Conv2d(
    int         inputSize,
    int         inputChannels,
    int         kernelSize,
    int         stride,
    std::string padding,
    int         numFilters,
    Activation  activation
)
    : inputSize(inputSize),
      inputChannels(inputChannels),
      kernelSize(kernelSize),
      stride(stride),
      numFilters(numFilters),
      activation(activation) {
    // Allocate memory for kernels

    if (padding == "SAME") {
        outputSize  = inputSize;
        paddingSize = ((stride - 1) * inputSize - stride + kernelSize) / 2;
    } else if (padding == "VALID") {
        paddingSize = 0;
        outputSize  = (inputSize - kernelSize) / stride + 1;
    }

    kernels.resize(kernelSize * kernelSize);
    initializeKernels();

    d_kernels = nullptr;

    CUDA_CHECK(
        hipMalloc((void**)&d_kernels, sizeof(float) * kernelSize * kernelSize)
    );
    toCuda();

    d_padded = nullptr;

    if (paddingSize > 0) {
        CUDA_CHECK(hipMalloc(
            (void**)&d_padded, sizeof(float) * (inputSize + 2 * paddingSize) *
                                   (inputSize + 2 * paddingSize) * inputChannels
        ));
    }
}

Layers::Conv2d::~Conv2d() {
    hipFree(d_kernels);
    hipFree(d_padded);
}

void Layers::Conv2d::initializeKernels() {
    std::fill(kernels.begin(), kernels.end(), 0.0f);
}

void Layers::Conv2d::setKernels(const std::vector<float>& kernels_input) {
    std::copy(kernels_input.begin(), kernels_input.end(), kernels.begin());
    toCuda();
}

void Layers::Conv2d::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_kernels, kernels.data(), sizeof(float) * kernelSize * kernelSize,
        hipMemcpyHostToDevice
    ));
}

void Layers::Conv2d::forward(const float* d_input, float* d_output) {
    // Padd input
    int THREADS_PER_BLOCK = 256;
    int BLOCKS =
        (outputSize * outputSize * inputChannels) / THREADS_PER_BLOCK + 1;

    pad_matrix_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(
        d_input, d_padded, inputSize, inputSize, inputChannels, paddingSize
    );

    // TODO: Implement 2D convolution
}

/*
Convolves input vector with kernel and stores result in output

input: matrix (inputSize + paddingSize) x (inputSize + paddingSize) x
inputChannels represented as a vector output: output matrix outputSize x
outputSize x numFilters

*/
void Layers::Conv2d::host_conv(const float* input, float* output) {
    // Iterate over output matrix
    for (int f = 0; f < numFilters; f++) {
        for (int i = 0; i < outputSize; i++) {
            for (int j = 0; j < outputSize; j++) {
                
                float sum = 0.0f;

                // Iterate over kernel and input matrix
                for (int k = 0; k < kernelSize; k++) {
                    for (int l = 0; l < kernelSize; l++) {
                        for (int c = 0; c < inputChannels; c++) {

                            // For now stride = 1
                            
                            int kernelIndex = k * (kernelSize * inputChannels * numFilters) + l * (inputChannels * numFilters) + c * (numFilters) + f;
                            int inputIndex  = (i * stride + k) * (inputSize * inputChannels) + (j + stride + l) * (inputChannels) + c;

                            sum += kernels[kernelIndex] * input[inputIndex];
                        }                      
                    }
                }

                output[i * (outputSize * numFilters) + j * (numFilters) + f] = sum;
            }
        }
    }
}