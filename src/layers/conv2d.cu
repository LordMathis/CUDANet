#include <string>
#include <iostream>

#include "activations.cuh"
#include "conv2d.cuh"
#include "convolution.cuh"
#include "cuda_helper.cuh"
#include "padding.cuh"

Layers::Conv2d::Conv2d(
    int         inputSize,
    int         inputChannels,
    int         kernelSize,
    int         stride,
    std::string padding,
    int         numFilters,
    Activation  activation
)
    : inputSize(inputSize),
      inputChannels(inputChannels),
      kernelSize(kernelSize),
      stride(stride),
      numFilters(numFilters),
      activation(activation) {
    // Allocate memory for kernels

    if (padding == "SAME") {
        outputSize  = inputSize;
        paddingSize = ((stride - 1) * inputSize - stride + kernelSize) / 2;
    } else if (padding == "VALID") {
        paddingSize = 0;
        outputSize  = (inputSize - kernelSize) / stride + 1;
    }

    kernels.resize(kernelSize * kernelSize * inputChannels * numFilters);
    initializeKernels();

    d_kernels = nullptr;

    CUDA_CHECK(
        hipMalloc((void**)&d_kernels, sizeof(float) * kernelSize * kernelSize * inputChannels * numFilters)
    );
    toCuda();

    d_padded = nullptr;

    CUDA_CHECK(hipMalloc(
        (void**)&d_padded, sizeof(float) * (inputSize + 2 * paddingSize) *
                                (inputSize + 2 * paddingSize) * inputChannels
    ));
}

Layers::Conv2d::~Conv2d() {
    hipFree(d_kernels);
    hipFree(d_padded);
}

void Layers::Conv2d::initializeKernels() {
    std::fill(kernels.begin(), kernels.end(), 0.0f);
}

void Layers::Conv2d::setKernels(const std::vector<float>& kernels_input) {
    std::copy(kernels_input.begin(), kernels_input.end(), kernels.begin());
    toCuda();
}

void Layers::Conv2d::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_kernels, kernels.data(), sizeof(float) * kernelSize * kernelSize * numFilters,
        hipMemcpyHostToDevice
    ));
}

void Layers::Conv2d::forward(const float* d_input, float* d_output) {
    // Pad input
    int THREADS_PER_BLOCK =  (inputSize + 2 * paddingSize) * (inputSize + 2 * paddingSize) * inputChannels;

    pad_matrix_kernel<<<1, THREADS_PER_BLOCK>>>(
        d_input, d_padded, inputSize, inputSize, inputChannels, paddingSize
    );

    // Convolve
    THREADS_PER_BLOCK = outputSize * outputSize * numFilters;
    convolution_kernel<<<1, THREADS_PER_BLOCK>>>(
        d_padded, d_kernels, d_output, inputSize + (2 * paddingSize), inputChannels, kernelSize, stride, numFilters, outputSize
    );

    CUDA_CHECK(hipDeviceSynchronize());
}

/*
Convolves input vector with kernel and stores result in output

input: matrix (inputSize + paddingSize) x (inputSize + paddingSize) x
inputChannels represented as a vector output: output matrix outputSize x
outputSize x numFilters

*/
void Layers::Conv2d::host_conv(const float* input, float* output) {
    // Iterate over output matrix
    for (int f = 0; f < numFilters; f++) {
        for (int i = 0; i < outputSize; i++) {
            for (int j = 0; j < outputSize; j++) {
                
                float sum = 0.0f;

                // Iterate over kernel and input matrix
                for (int k = 0; k < kernelSize; k++) {
                    for (int l = 0; l < kernelSize; l++) {
                        for (int c = 0; c < inputChannels; c++) {
                            
                            int kernelIndex = k * (kernelSize * inputChannels * numFilters) + l * (inputChannels * numFilters) + c * (numFilters) + f;
                            int inputIndex  = (i * stride + k) * (inputSize * inputChannels) + (j * stride + l) * (inputChannels) + c;

                            sum += kernels[kernelIndex] * input[inputIndex];
                        }                      
                    }
                }

                output[i * (outputSize * numFilters) + j * (numFilters) + f] = sum;
            }
        }
    }
}