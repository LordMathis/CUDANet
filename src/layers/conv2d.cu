#include <iostream>
#include <string>

#include "activations.cuh"
#include "conv2d.cuh"
#include "convolution.cuh"
#include "cuda_helper.cuh"
#include "matrix_math.cuh"
#include "padding.cuh"

Layers::Conv2d::Conv2d(
    int         inputSize,
    int         inputChannels,
    int         kernelSize,
    int         stride,
    Padding padding,
    int         numFilters,
    Activation  activation
)
    : inputSize(inputSize),
      inputChannels(inputChannels),
      kernelSize(kernelSize),
      stride(stride),
      numFilters(numFilters),
      activation(activation) {
    // Allocate memory for kernels

    switch (padding)
    {
    case SAME:
        outputSize  = inputSize;
        paddingSize = ((stride - 1) * inputSize - stride + kernelSize) / 2;
        break;

    case VALID:
        paddingSize = 0;
        outputSize  = (inputSize - kernelSize) / stride + 1;
        break;
    
    default:
        break;
    }

    kernels.resize(kernelSize * kernelSize * inputChannels * numFilters);
    initializeKernels();

    d_kernels = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_kernels,
        sizeof(float) * kernelSize * kernelSize * inputChannels * numFilters
    ));

    biases.resize(outputSize * outputSize * numFilters);
    initializeBiases();

    d_biases = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_biases, sizeof(float) * outputSize * outputSize * numFilters
    ));

    d_padded = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_padded, sizeof(float) * (inputSize + 2 * paddingSize) *
                               (inputSize + 2 * paddingSize) * inputChannels
    ));

    toCuda();
}

Layers::Conv2d::~Conv2d() {
    hipFree(d_kernels);
    hipFree(d_biases);
    hipFree(d_padded);
}

void Layers::Conv2d::initializeKernels() {
    std::fill(kernels.begin(), kernels.end(), 0.0f);
}

void Layers::Conv2d::initializeBiases() {
    std::fill(biases.begin(), biases.end(), 0.0f);
}

void Layers::Conv2d::setKernels(const std::vector<float>& kernels_input) {
    std::copy(kernels_input.begin(), kernels_input.end(), kernels.begin());
    toCuda();
}

void Layers::Conv2d::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_kernels, kernels.data(),
        sizeof(float) * kernelSize * kernelSize * inputChannels * numFilters,
        hipMemcpyHostToDevice
    ));

    CUDA_CHECK(hipMemcpy(
        d_biases, biases.data(),
        sizeof(float) * outputSize * outputSize * numFilters,
        hipMemcpyHostToDevice
    ));
}

void Layers::Conv2d::forward(const float* d_input, float* d_output) {
    // Pad input
    int THREADS_PER_BLOCK = (inputSize + 2 * paddingSize) *
                            (inputSize + 2 * paddingSize) * inputChannels;

    pad_matrix_kernel<<<1, THREADS_PER_BLOCK>>>(
        d_input, d_padded, inputSize, inputSize, inputChannels, paddingSize
    );

    // Convolve
    THREADS_PER_BLOCK = outputSize * outputSize * numFilters;
    convolution_kernel<<<1, THREADS_PER_BLOCK>>>(
        d_padded, d_kernels, d_output, inputSize + (2 * paddingSize),
        inputChannels, kernelSize, stride, numFilters, outputSize
    );

    // Add bias
    vec_vec_add_kernel<<<1, biases.size()>>>(
        d_biases, d_output, d_output, biases.size()
    );

    CUDA_CHECK(hipDeviceSynchronize());
}

/*
Convolves input vector with kernel and stores result in output

input: matrix (inputSize + paddingSize) x (inputSize + paddingSize) x
inputChannels represented as a vector output: output matrix outputSize x
outputSize x numFilters

*/
void Layers::Conv2d::host_conv(const float* input, float* output) {
    // Iterate over output matrix
    for (int tid = 0; tid < outputSize * outputSize * numFilters; tid++)
    {  
        // Get output index
        int f = tid / (outputSize * outputSize);
        int i = tid % (outputSize * outputSize) / outputSize;
        int j = tid % outputSize;

        float sum = 0.0f;

        // Iterate over kernel and input matrix
        for (int k = 0; k < kernelSize; k++) {
            for (int l = 0; l < kernelSize; l++) {
                for (int c = 0; c < inputChannels; c++) {
                    int kernelIndex =
                        f * kernelSize * kernelSize * inputChannels +
                        c * kernelSize * kernelSize + k * kernelSize +
                        l;
                    int inputIndex = c * inputSize * inputSize +
                                        (i * stride + k) * inputSize +
                                        (j * stride + l);

                    sum += kernels[kernelIndex] * input[inputIndex];
                }
            }
        }

        int outputIndex =
            f * outputSize * outputSize + i * outputSize + j;

        output[outputIndex] = sum;
    }
}