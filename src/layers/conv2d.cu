#include <iostream>
#include <vector>

#include "activation.cuh"
#include "conv2d.cuh"
#include "convolution.cuh"
#include "cuda_helper.cuh"
#include "layer.cuh"
#include "matmul.cuh"
#include "vector.cuh"

using namespace CUDANet::Layers;

Conv2d::Conv2d(
    shape2d          inputSize,
    int            inputChannels,
    shape2d          kernelSize,
    shape2d          stride,
    int            numFilters,
    shape2d          paddingSize,
    ActivationType activationType
)
    : inputSize(inputSize),
      inputChannels(inputChannels),
      kernelSize(kernelSize),
      stride(stride),
      numFilters(numFilters),
      paddingSize(paddingSize) {

    outputSize = {
        (inputSize.first - kernelSize.first + 2 * paddingSize.first) /
                stride.first + 1,
        (inputSize.second - kernelSize.second + 2 * paddingSize.second) /
                stride.second + 1
    };

    activation =
        new Activation(activationType, outputSize.first * outputSize.second * numFilters);

    d_output = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_output, sizeof(float) * outputSize.first * outputSize.second * numFilters
    ));

    weights.resize(kernelSize.first * kernelSize.second * inputChannels * numFilters);
    initializeWeights();

    d_weights = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_weights,
        sizeof(float) * kernelSize.first * kernelSize.second * inputChannels * numFilters
    ));

    biases.resize(numFilters);
    initializeBiases();

    d_biases = nullptr;
    CUDA_CHECK(hipMalloc((void**)&d_biases, sizeof(float) * numFilters));

    toCuda();
}

Conv2d::~Conv2d() {
    hipFree(d_output);
    hipFree(d_weights);
    hipFree(d_biases);
    delete activation;
}

void Conv2d::initializeWeights() {
    std::fill(weights.begin(), weights.end(), 0.0f);
}

void Conv2d::initializeBiases() {
    std::fill(biases.begin(), biases.end(), 0.0f);
}

void Conv2d::setWeights(const float* weights_input) {
    std::copy(weights_input, weights_input + weights.size(), weights.begin());
    toCuda();
}

std::vector<float> Conv2d::getWeights() {
    return weights;
}

void Conv2d::setBiases(const float* biases_input) {
    std::copy(biases_input, biases_input + biases.size(), biases.begin());
    toCuda();
}

std::vector<float> Conv2d::getBiases() {
    return biases;
}

void Conv2d::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_weights, weights.data(),
        sizeof(float) * kernelSize.first * kernelSize.second * inputChannels * numFilters,
        hipMemcpyHostToDevice
    ));

    CUDA_CHECK(hipMemcpy(
        d_biases, biases.data(), sizeof(float) * numFilters,
        hipMemcpyHostToDevice
    ));
}

float* Conv2d::forward(const float* d_input) {
    // Convolve
    dim3 block(8, 8, 8);
    dim3 grid(
        (outputSize.first + block.x - 1) / block.x,
        (outputSize.second + block.y - 1) / block.y,
        (numFilters + block.z - 1) / block.z
    );

    CUDANet::Utils::clear(d_output, outputSize.first * outputSize.second * numFilters);

    Kernels::convolution<<<grid, block>>>(
        d_input, d_weights, d_biases, d_output, inputSize, inputChannels,
        paddingSize, kernelSize, stride, numFilters, outputSize
    );
    CUDA_CHECK(hipGetLastError());

    // Apply activation
    activation->activate(d_output);

    CUDA_CHECK(hipDeviceSynchronize());

    return d_output;
}

int Conv2d::getOutputSize() {
    return outputSize.first * outputSize.second * numFilters;
}

int Conv2d::getInputSize() {
    return inputSize.first * inputSize.second * inputChannels;
}

shape2d Conv2d::getOutputDims() {
    return outputSize;
}