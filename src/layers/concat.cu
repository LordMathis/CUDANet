#include "concat.cuh"
#include "cuda_helper.cuh"

using namespace CUDANet;


Layers::Concat::Concat(const unsigned int inputASize, const unsigned int inputBSize)
    : inputASize(inputASize), inputBSize(inputBSize) {

    d_output = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_output, sizeof(float) * (inputASize + inputBSize)
    ));

}

Layers::Concat::~Concat() {
    hipFree(d_output);
}


float* Layers::Concat::forward(const float* d_input_A, const float* d_input_B) {
    CUDA_CHECK(hipMemcpy(
        d_output, d_input_A, sizeof(float) * inputASize, hipMemcpyDeviceToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        d_output + inputASize, d_input_B,
        sizeof(float) * inputBSize, hipMemcpyDeviceToDevice
    ));

    return d_output;
}
