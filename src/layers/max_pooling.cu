#include "cuda_helper.cuh"
#include "max_pooling.cuh"
#include "pooling.cuh"

using namespace CUDANet::Layers;

MaxPooling2d::MaxPooling2d(
    dim2d          inputSize,
    int            nChannels,
    dim2d          poolingSize,
    dim2d          stride,
    dim2d          padding,
    ActivationType activationType
)
    : inputSize(inputSize),
      nChannels(nChannels),
      poolingSize(poolingSize),
      stride(stride),
      padding(padding) {
    outputSize = {
        (inputSize.first + 2 * padding.first - poolingSize.first) /
                stride.first +
            1,
        (inputSize.second + 2 * padding.second - poolingSize.second) /
                stride.second +
            1
    };

    activation = new Activation(
        activationType, outputSize.first * outputSize.second * nChannels
    );

    d_output = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_output,
        sizeof(float) * outputSize.first * outputSize.second * nChannels
    ));
}

MaxPooling2d::~MaxPooling2d() {
    hipFree(d_output);
    delete activation;
}

float* MaxPooling2d::forward(const float* d_input) {
    dim3 block(8, 8, 8);
    dim3 grid(
        (outputSize.first + block.x - 1) / block.x,
        (outputSize.second + block.y - 1) / block.y,
        (nChannels + block.z - 1) / block.z
    );

    Kernels::max_pooling<<<grid, block>>>(
        d_input, d_output, inputSize, outputSize, nChannels, poolingSize,
        stride, padding
    );
    CUDA_CHECK(hipGetLastError());

    activation->activate(d_output);
    CUDA_CHECK(hipDeviceSynchronize());

    return d_output;
}

int MaxPooling2d::getOutputSize() {
    return outputSize.first * outputSize.second * nChannels;
}

int MaxPooling2d::getInputSize() {
    return inputSize.first * inputSize.second * nChannels;
}

dim2d MaxPooling2d::getOutputDims() {
    return outputSize;
}