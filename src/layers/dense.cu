#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <functional>
#include <iostream>

#include "activations.cuh"
#include "cuda_helper.cuh"
#include "dense.cuh"
#include "matrix_math.cuh"

Layers::Dense::Dense(int inputSize, int outputSize, Activation activation)
    : inputSize(inputSize), outputSize(outputSize), activation(activation) {
    // Allocate memory for weights and biases
    weights.resize(outputSize * inputSize);
    biases.resize(outputSize);

    initializeWeights();
    initializeBiases();

    d_weights = nullptr;
    d_biases  = nullptr;

    // Allocate GPU memory for weights and biases
    CUDA_CHECK(
        hipMalloc((void**)&d_weights, sizeof(float) * inputSize * outputSize)
    );
    CUDA_CHECK(hipMalloc((void**)&d_biases, sizeof(float) * outputSize));

    toCuda();
}

Layers::Dense::~Dense() {
    // Free GPU memory
    hipFree(d_weights);
    hipFree(d_biases);
}

void Layers::Dense::initializeWeights() {
    std::fill(weights.begin(), weights.end(), 0.0f);
}

void Layers::Dense::initializeBiases() {
    std::fill(biases.begin(), biases.end(), 0.0f);
}

void Layers::Dense::forward(const float* d_input, float* d_output) {
    mat_vec_mul_kernel<<<1, outputSize>>>(
        d_weights, d_input, d_output, inputSize, outputSize
    );

    vec_vec_add_kernel<<<1, outputSize>>>(
        d_biases, d_output, d_output, outputSize
    );

    switch (activation) {
        case SIGMOID:
            sigmoid_kernel<<<1, outputSize>>>(d_output, d_output, outputSize);
            break;

        case RELU:
            relu_kernel<<<1, outputSize>>>(d_output, d_output, outputSize);
            break;

        default:
            break;
    }

    CUDA_CHECK(hipDeviceSynchronize());
}

void Layers::Dense::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_weights, weights.data(), sizeof(float) * inputSize * outputSize,
        hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        d_biases, biases.data(), sizeof(float) * outputSize,
        hipMemcpyHostToDevice
    ));
}

void Layers::Dense::setWeights(const float* weights_input) {
    std::copy(weights_input, weights_input + weights.size(), weights.begin());
    toCuda();
}

void Layers::Dense::setBiases(const float* biases_input) {
    std::copy(biases_input, biases_input + biases.size(), biases.begin());
    toCuda();
}