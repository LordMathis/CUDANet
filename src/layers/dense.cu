#include "dense.cuh"
#include "cuda_helper.cuh"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdio>
#include <iostream>

Layers::Dense::Dense(int inputSize, int outputSize, hipblasHandle_t cublasHandle)
    : inputSize(inputSize), outputSize(outputSize), cublasHandle(cublasHandle) {

    // Allocate memory for weights and biases
    weights.resize(outputSize * inputSize);
    biases.resize(outputSize);

    initializeWeights();
    initializeBiases();

    d_weights = nullptr;
    d_biases = nullptr;

    // Allocate GPU memory for weights and biases
    CUDA_CHECK(hipMalloc((void**)&d_weights, sizeof(float) * inputSize * outputSize));
    CUDA_CHECK(hipMalloc((void**)&d_biases, sizeof(float) * outputSize));

    toCuda();
}

Layers::Dense::~Dense() {
    // Free GPU memory
    hipFree(d_weights);
    hipFree(d_biases);
}

void Layers::Dense::initializeWeights() {

    for (int j = 0; j < inputSize; ++j) {
        for (int i = 0; i < outputSize; ++i) {
            int idx = IDX2C(i, j, outputSize);
            weights[idx] = 0.0f;
        }
    }
}

void Layers::Dense::initializeBiases() {
    std::fill(biases.begin(), biases.end(), 0.0f);
}

void Layers::Dense::forward(const float* d_input, float* d_output) {
    const float alpha = 1.0f;
    const float beta = 1.0f;

    CUBLAS_CHECK(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, inputSize, outputSize, &alpha, d_weights, inputSize, d_input, 1, &beta, d_output, 1));
    CUBLAS_CHECK(hipblasSaxpy(cublasHandle, outputSize, &alpha, d_biases, 1, d_output, 1));
}

void Layers::Dense::toCuda() {
    CUBLAS_CHECK(hipblasSetMatrix(outputSize, inputSize, sizeof(float), weights.data(), outputSize, d_weights, outputSize));
    CUBLAS_CHECK(hipblasSetVector(biases.size(), sizeof(float), biases.data(), 1, d_biases, 1));
}

void Layers::Dense::setWeights(const std::vector<std::vector<float>>& weights_input) {
    int numWeights = inputSize * outputSize;

    if (weights.size() != numWeights) {
        std::cerr << "Invalid number of weights" << std::endl;
        exit(EXIT_FAILURE);
    }

    for (int j = 0; j < inputSize; ++j) {
        for (int i = 0; i < outputSize; ++i) {
            int idx = IDX2C(i, j, outputSize);
            weights[idx] = weights_input[i][j];
        }
    }

    toCuda();
}

void Layers::Dense::setBiases(const std::vector<float>& biases_input) {
    std::copy(biases_input.begin(), biases_input.end(), biases.begin());
    toCuda();
}