#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <functional>
#include <iostream>

#include "activations.cuh"
#include "cuda_helper.cuh"
#include "dense.cuh"
#include "matmul.cuh"

Layers::Dense::Dense(int inputSize, int outputSize, Layers::Activation activation)
    : inputSize(inputSize), outputSize(outputSize), activation(activation) {
    // Allocate memory for weights and biases
    weights.resize(outputSize * inputSize);
    biases.resize(outputSize);

    initializeWeights();
    initializeBiases();

    d_weights = nullptr;
    d_biases  = nullptr;

    // Allocate GPU memory for weights and biases
    CUDA_CHECK(
        hipMalloc((void**)&d_weights, sizeof(float) * inputSize * outputSize)
    );
    CUDA_CHECK(hipMalloc((void**)&d_biases, sizeof(float) * outputSize));

    toCuda();
}

Layers::Dense::~Dense() {
    // Free GPU memory
    hipFree(d_weights);
    hipFree(d_biases);
}

void Layers::Dense::initializeWeights() {
    std::fill(weights.begin(), weights.end(), 0.0f);
}

void Layers::Dense::initializeBiases() {
    std::fill(biases.begin(), biases.end(), 0.0f);
}

void Layers::Dense::forward(const float* d_input, float* d_output) {
    Kernels::mat_vec_mul<<<1, outputSize>>>(
        d_weights, d_input, d_output, inputSize, outputSize
    );

    Kernels::vec_vec_add<<<1, outputSize>>>(
        d_biases, d_output, d_output, outputSize
    );

    switch (activation) {
        case SIGMOID:
            Kernels::sigmoid<<<1, outputSize>>>(d_output, d_output, outputSize);
            break;

        case RELU:
            Kernels::relu<<<1, outputSize>>>(d_output, d_output, outputSize);
            break;

        default:
            break;
    }

    CUDA_CHECK(hipDeviceSynchronize());
}

void Layers::Dense::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_weights, weights.data(), sizeof(float) * inputSize * outputSize,
        hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        d_biases, biases.data(), sizeof(float) * outputSize,
        hipMemcpyHostToDevice
    ));
}

void Layers::Dense::setWeights(const float* weights_input) {
    std::copy(weights_input, weights_input + weights.size(), weights.begin());
    toCuda();
}

void Layers::Dense::setBiases(const float* biases_input) {
    std::copy(biases_input, biases_input + biases.size(), biases.begin());
    toCuda();
}