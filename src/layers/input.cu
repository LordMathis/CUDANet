#include "cuda_helper.cuh"
#include "input.cuh"

using namespace CUDANet::Layers;

Input::Input(int inputSize) : inputSize(inputSize) {
    d_output = nullptr;
    CUDA_CHECK(hipMalloc((void**)&d_output, sizeof(float) * inputSize));
}

Input::~Input() {
    hipFree(d_output);
}

float* Input::forward(const float* input) {
    CUDA_CHECK(hipMemcpy(
        d_output, input, sizeof(float) * inputSize, hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipDeviceSynchronize());

    return d_output;
}
