#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "activation.cuh"
#include "activation_functions.cuh"
#include "cuda_helper.cuh"
#include "matmul.cuh"
#include "vector.cuh"

using namespace CUDANet::Layers;

Activation::Activation(ActivationType activation, const int length)
    : activationType(activation), length(length) {
    if (activationType == SOFTMAX) {
        d_max = nullptr;
        CUDA_CHECK(hipMalloc((void**)&d_max, sizeof(float) * length));

        d_softmax_sum = nullptr;
        CUDA_CHECK(hipMalloc((void**)&d_softmax_sum, sizeof(float) * length));
    }

    gridSize = (length + BLOCK_SIZE - 1) / BLOCK_SIZE;
}

Activation::~Activation() {
    if (activationType == SOFTMAX) {
        hipFree(d_softmax_sum);
        hipFree(d_max);
    }
}

void Activation::activate(float* d_input) {

    // float sum = 0.0f;

    switch (activationType) {
        case SIGMOID:
            Kernels::sigmoid<<<gridSize, BLOCK_SIZE>>>(
                d_input, d_input, length
            );
            CUDA_CHECK(hipGetLastError());
            break;

        case RELU:
            Kernels::relu<<<gridSize, BLOCK_SIZE>>>(d_input, d_input, length);
            CUDA_CHECK(hipGetLastError());
            break;
        case SOFTMAX:

            // Find max value
            Utils::max(d_input, d_max, length);

            // Subtract max value to improve numerical stability
            Kernels::vec_scalar_sub<<<gridSize, BLOCK_SIZE>>>(
                d_input, d_input, d_max, length
            );
            CUDA_CHECK(hipGetLastError());

            // Compute exponentials
            Kernels::vec_exp<<<gridSize, BLOCK_SIZE>>>(
                d_input, d_input, length
            );
            CUDA_CHECK(hipGetLastError());

            // Find sum
            Utils::sum(d_input, d_softmax_sum, length);

            Kernels::vec_scalar_div<<<gridSize, BLOCK_SIZE>>>(
                d_input, d_input, d_softmax_sum, length
            );
            CUDA_CHECK(hipGetLastError());

            break;

        default:
            break;    
    }

    CUDA_CHECK(hipDeviceSynchronize());
}

