#include <vector>

#include "activation.cuh"
#include "batch_norm.cuh"
#include "cuda_helper.cuh"
#include "layer.cuh"
#include "matmul.cuh"
#include "vector.cuh"

using namespace CUDANet::Layers;

BatchNorm2d::BatchNorm2d(
    dim2d          inputSize,
    int            inputChannels,
    float          epsilon,
    ActivationType activationType
)
    : inputSize(inputSize), inputChannels(inputChannels) {
    activation = new Activation(
        activationType, inputSize.first * inputSize.second * inputChannels
    );

    d_output = nullptr;
    CUDA_CHECK(hipMalloc(
        (void **)&d_output,
        sizeof(float) * inputSize.first * inputSize.second * inputChannels
    ));

    d_mean = nullptr;
    CUDA_CHECK(hipMalloc(
        (void **)&d_mean, sizeof(float) * inputSize.first * inputSize.second
    ));

    d_mean_sub = nullptr;
    CUDA_CHECK(hipMalloc(
        (void **)&d_mean_sub, sizeof(float) * inputSize.first * inputSize.second
    ));

    d_sqrt_var = nullptr;
    CUDA_CHECK(hipMalloc(
        (void **)&d_sqrt_var, sizeof(float) * inputSize.first * inputSize.second
    ));

    d_weights = nullptr;
    CUDA_CHECK(hipMalloc((void **)&d_weights, sizeof(float) * inputChannels));

    d_biases = nullptr;
    CUDA_CHECK(hipMalloc((void **)&d_biases, sizeof(float) * inputChannels));

    d_length     = nullptr;
    float length = (float)inputSize.first * inputSize.second;
    CUDA_CHECK(hipMalloc((void **)&d_length, sizeof(float)));
    CUDA_CHECK(
        hipMemcpy(d_length, &length, sizeof(float), hipMemcpyHostToDevice)
    );

    d_epsilon = nullptr;
    CUDA_CHECK(hipMalloc((void **)&d_epsilon, sizeof(float)));
    CUDA_CHECK(
        hipMemcpy(d_epsilon, &epsilon, sizeof(float), hipMemcpyHostToDevice)
    );

    weights.resize(inputChannels);
    biases.resize(inputChannels);

    initializeWeights();
    initializeBiases();

    toCuda();

    gridSize =
        (inputSize.first * inputSize.second + BLOCK_SIZE - 1) / BLOCK_SIZE;
}

BatchNorm2d::~BatchNorm2d() {
    hipFree(d_output);
    hipFree(d_mean);
    hipFree(d_mean_sub);
    hipFree(d_sqrt_var);
    hipFree(d_weights);
    hipFree(d_biases);
    hipFree(d_length);
    hipFree(d_epsilon);
}

void BatchNorm2d::initializeWeights() {
    std::fill(weights.begin(), weights.end(), 1.0f);
}

void BatchNorm2d::initializeBiases() {
    std::fill(biases.begin(), biases.end(), 0.0f);
}

void BatchNorm2d::setWeights(const float *weights_input) {
    std::copy(weights_input, weights_input + weights.size(), weights.begin());
    toCuda();
}

std::vector<float> BatchNorm2d::getWeights() {
    return weights;
}

void BatchNorm2d::setBiases(const float *biases_input) {
    std::copy(biases_input, biases_input + biases.size(), biases.begin());
    toCuda();
}

std::vector<float> BatchNorm2d::getBiases() {
    return biases;
}

void BatchNorm2d::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_weights, weights.data(), sizeof(float) * inputChannels,
        hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        d_biases, biases.data(), sizeof(float) * inputChannels,
        hipMemcpyHostToDevice
    ));
}

int BatchNorm2d::getInputSize() {
    return inputSize.first * inputSize.second * inputChannels;
}

int BatchNorm2d::getOutputSize() {
    return inputSize.first * inputSize.second * inputChannels;
}

dim2d BatchNorm2d::getOutputDims() {
    return inputSize;
}

float *BatchNorm2d::forward(const float *d_input) {
    // Compute per-channel batch normalization
    for (int i = 0; i < inputChannels; i++) {
        // Compute mean
        Utils::mean(
            d_input + i * inputSize.first * inputSize.second, d_mean, d_length,
            inputSize.first * inputSize.second
        );

        // Subtract mean from input
        Kernels::vec_scalar_sub<<<gridSize, BLOCK_SIZE>>>(
            d_input + i * inputSize.first * inputSize.second, d_mean_sub,
            &d_mean[0], inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());

        // Compute variance
        Utils::var(
            d_mean_sub, d_sqrt_var, d_length, inputSize.first * inputSize.second
        );

        // Add epsilon to variance to avoid division by zero
        Kernels::vec_scalar_add<<<gridSize, BLOCK_SIZE>>>(
            d_sqrt_var, d_sqrt_var, &d_epsilon[0],
            inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());

        // Compute squared root of variance
        Kernels::vec_sqrt<<<gridSize, BLOCK_SIZE>>>(
            d_sqrt_var, d_sqrt_var, inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());

        // Divide by squared root of variance
        Kernels::vec_scalar_div<<<gridSize, BLOCK_SIZE>>>(
            d_mean_sub, d_output + i * inputSize.first * inputSize.second,
            &d_sqrt_var[0], inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());

        // Multiply by weights
        Kernels::vec_scalar_mul<<<gridSize, BLOCK_SIZE>>>(
            d_output + i * inputSize.first * inputSize.second,
            d_output + i * inputSize.first * inputSize.second, &d_weights[i],
            inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());

        // Add biases
        Kernels::vec_scalar_add<<<gridSize, BLOCK_SIZE>>>(
            d_output + i * inputSize.first * inputSize.second,
            d_output + i * inputSize.first * inputSize.second, &d_biases[i],
            inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());
    }

    return d_output;
}