#include "cuda_helper.cuh"
#include "input.hpp"

using namespace CUDANet::Layers;

void Input::initCUDA() {
    d_output = nullptr;
    CUDA_CHECK(hipMalloc((void**)&d_output, sizeof(float) * inputSize));
}

void Input::delCUDA() {
    hipFree(d_output);
}

float* Input::forwardCUDA(const float* input) {
    CUDA_CHECK(hipMemcpy(
        d_output, input, sizeof(float) * inputSize, hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipDeviceSynchronize());

    return d_output;
}