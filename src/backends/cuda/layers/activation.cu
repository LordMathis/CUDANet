#include "hip/hip_runtime.h"
#include <vector>

#include "activation.hpp"
#include "activation_functions.cuh"
#include "cuda_helper.cuh"
#include "matmul.cuh"
#include "vector.cuh"

using namespace CUDANet::Layers;

void Activation::initCUDA() {
    if (activationType == SOFTMAX) {
        d_softmax_sum = nullptr;
        CUDA_CHECK(hipMalloc((void**)&d_softmax_sum, sizeof(float) * length));

        d_max = nullptr;
        CUDA_CHECK(hipMalloc((void**)&d_max, sizeof(float) * length));
    }

    gridSize = (length + BLOCK_SIZE - 1) / BLOCK_SIZE;
}

void Activation::delCUDA() {
    if (activationType == SOFTMAX) {
        CUDA_CHECK(hipFree(d_softmax_sum));
        CUDA_CHECK(hipFree(d_max));
    }
}

void Activation::activateCUDA(float* d_input) {

    // float sum = 0.0f;

    switch (activationType) {
        case SIGMOID:
            Kernels::sigmoid<<<gridSize, BLOCK_SIZE>>>(
                d_input, d_input, length
            );
            CUDA_CHECK(hipGetLastError());
            break;

        case RELU:
            Kernels::relu<<<gridSize, BLOCK_SIZE>>>(d_input, d_input, length);
            CUDA_CHECK(hipGetLastError());
            break;
        case SOFTMAX:

            // Find max value
            Utils::max(d_input, d_max, length);

            // Subtract max value to improve numerical stability
            Kernels::vec_scalar_sub<<<gridSize, BLOCK_SIZE>>>(
                d_input, d_input, &d_max[0], length
            );
            CUDA_CHECK(hipGetLastError());

            // Compute exponentials
            Kernels::vec_exp<<<gridSize, BLOCK_SIZE>>>(
                d_input, d_input, length
            );
            CUDA_CHECK(hipGetLastError());
            
            // Find sum
            Utils::sum(d_input, d_softmax_sum, length);

            Kernels::vec_scalar_div<<<gridSize, BLOCK_SIZE>>>(
                d_input, d_input, &d_softmax_sum[0], length
            );
            CUDA_CHECK(hipGetLastError());
            break;

        default:
            break;    
    }

    CUDA_CHECK(hipDeviceSynchronize());
}
