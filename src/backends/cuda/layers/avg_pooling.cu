#include "avg_pooling.hpp"
#include "cuda_helper.cuh"
#include "pooling.cuh"

using namespace CUDANet::Layers;

void AvgPooling2d::initCUDA() {
    d_output = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_output,
        sizeof(float) * outputSize.first * outputSize.second * nChannels
    ));
}

void AvgPooling2d::delCUDA() {
    hipFree(d_output);
}

float* AvgPooling2d::forwardCUDA(const float* d_input) {
    dim3 block(8, 8, 8);
    dim3 grid(
        (outputSize.first + block.x - 1) / block.x,
        (outputSize.second + block.y - 1) / block.y,
        (nChannels + block.z - 1) / block.z
    );

    Kernels::avg_pooling<<<grid, block>>>(
        d_input, d_output, inputSize, outputSize, nChannels, poolingSize,
        stride, padding
    );
    CUDA_CHECK(hipGetLastError());

    activation->activate(d_output);
    CUDA_CHECK(hipDeviceSynchronize());

    return d_output;
}

void AdaptiveAvgPooling2d::initCUDA() {
    hipFree(d_output);
    hipMalloc(
        (void**)&d_output,
        sizeof(float) * outputSize.first * outputSize.second * nChannels
    );
}
