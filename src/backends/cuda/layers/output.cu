#include "output.hpp"

#include "cuda_helper.cuh"

using namespace CUDANet::Layers;

float* Output::forwardCUDA(const float* input) {
    CUDA_CHECK(hipMemcpy(
        h_output, input, sizeof(float) * inputSize, hipMemcpyDeviceToHost
    ));
    CUDA_CHECK(hipDeviceSynchronize());

    return h_output;
}