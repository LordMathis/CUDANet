#include <vector>

#include "activation.hpp"
#include "conv2d.hpp"
#include "convolution.cuh"
#include "cuda_helper.cuh"
#include "layer.hpp"
#include "matmul.cuh"
#include "vector.cuh"

using namespace CUDANet::Layers;

void Conv2d::initCUDA() {
    d_output = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_output,
        sizeof(float) * outputSize.first * outputSize.second * numFilters
    ));

    d_weights = nullptr;
    CUDA_CHECK(hipMalloc(
        (void**)&d_weights, sizeof(float) * kernelSize.first *
                                kernelSize.second * inputChannels * numFilters
    ));

    d_biases = nullptr;
    CUDA_CHECK(hipMalloc((void**)&d_biases, sizeof(float) * numFilters));
}

void Conv2d::delCUDA() {
    hipFree(d_output);
    hipFree(d_weights);
    hipFree(d_biases);
}

void Conv2d::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_weights, weights.data(),
        sizeof(float) * kernelSize.first * kernelSize.second * inputChannels *
            numFilters,
        hipMemcpyHostToDevice
    ));

    CUDA_CHECK(hipMemcpy(
        d_biases, biases.data(), sizeof(float) * numFilters,
        hipMemcpyHostToDevice
    ));
}

float* Conv2d::forwardCUDA(const float* d_input) {
    // Convolve
    dim3 block(8, 8, 8);
    dim3 grid(
        (outputSize.first + block.x - 1) / block.x,
        (outputSize.second + block.y - 1) / block.y,
        (numFilters + block.z - 1) / block.z
    );

    CUDANet::Utils::clear(d_output, outputSize.first * outputSize.second * numFilters);

    Kernels::convolution<<<grid, block>>>(
        d_input, d_weights, d_biases, d_output, inputSize, inputChannels,
        paddingSize, kernelSize, stride, numFilters, outputSize
    );
    CUDA_CHECK(hipGetLastError());

    // Apply activation
    activation->activate(d_output);

    CUDA_CHECK(hipDeviceSynchronize());

    return d_output;
}
