#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <functional>
#include <iostream>

#include "vector.cuh"
#include "activation.hpp"
#include "cuda_helper.cuh"
#include "dense.hpp"
#include "matmul.cuh"

using namespace CUDANet::Layers;

void Dense::initCUDA() {
    d_output = nullptr;

    CUDA_CHECK(hipMalloc((void**)&d_output, sizeof(float) * outputSize));

    d_weights = nullptr;
    d_biases  = nullptr;

    // Allocate GPU memory for weights and biases
    CUDA_CHECK(
        hipMalloc((void**)&d_weights, sizeof(float) * inputSize * outputSize)
    );
    CUDA_CHECK(hipMalloc((void**)&d_biases, sizeof(float) * outputSize));
    toCuda();

    // Calculate block and grid sizes
    forwardGridSize =
        (std::max(inputSize, outputSize) + BLOCK_SIZE - 1) / BLOCK_SIZE;
    biasGridSize = (outputSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
}

void Dense::delCUDA() {
    hipFree(d_output);
    hipFree(d_weights);
    hipFree(d_biases);
}

void Dense::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_weights, weights.data(), sizeof(float) * inputSize * outputSize,
        hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        d_biases, biases.data(), sizeof(float) * outputSize,
        hipMemcpyHostToDevice
    ));
}

float* Dense::forwardCUDA(const float* d_input) {
    Kernels::mat_vec_mul<<<forwardGridSize, BLOCK_SIZE>>>(
        d_weights, d_input, d_output, inputSize, outputSize
    );
    CUDA_CHECK(hipGetLastError());

    Kernels::vec_vec_add<<<biasGridSize, BLOCK_SIZE>>>(
        d_biases, d_output, d_output, outputSize
    );
    CUDA_CHECK(hipGetLastError());

    activation->activate(d_output);
    CUDA_CHECK(hipDeviceSynchronize());

    return d_output;
}
