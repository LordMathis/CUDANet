#include "concat.hpp"
#include "cuda_helper.cuh"

using namespace CUDANet::Layers;

void Concat::initCUDA() {
    d_output = nullptr;
    CUDA_CHECK(
        hipMalloc((void**)&d_output, sizeof(float) * (inputASize + inputBSize))
    );
}

void Concat::delCUDA() {
    hipFree(d_output);
}

float* Concat::forwardCUDA(const float* d_input_A, const float* d_input_B) {
    CUDA_CHECK(hipMemcpy(
        d_output, d_input_A, sizeof(float) * inputASize,
        hipMemcpyDeviceToDevice
    ));

    CUDA_CHECK(hipMemcpy(
        d_output + inputASize, d_input_B, sizeof(float) * inputBSize,
        hipMemcpyDeviceToDevice
    ));

    CUDA_CHECK(hipDeviceSynchronize());

    return d_output;
}