#include "hip/hip_runtime.h"
#include <vector>

#include "activation.hpp"
#include "batch_norm.hpp"
#include "cuda_helper.cuh"
#include "layer.hpp"
#include "matmul.cuh"
#include "vector.cuh"

using namespace CUDANet::Layers;

void BatchNorm2d::initCUDA() {
    d_output = nullptr;
    CUDA_CHECK(hipMalloc(
        (void **)&d_output,
        sizeof(float) * inputSize.first * inputSize.second * inputChannels
    ));

    d_running_mean = nullptr;
    CUDA_CHECK(
        hipMalloc((void **)&d_running_mean, sizeof(float) * inputChannels)
    );

    d_running_var = nullptr;
    CUDA_CHECK(
        hipMalloc((void **)&d_running_var, sizeof(float) * inputChannels)
    );

    d_weights = nullptr;
    CUDA_CHECK(hipMalloc((void **)&d_weights, sizeof(float) * inputChannels));

    d_biases = nullptr;
    CUDA_CHECK(hipMalloc((void **)&d_biases, sizeof(float) * inputChannels));

    d_length     = nullptr;
    float length = (float)inputSize.first * inputSize.second;
    CUDA_CHECK(hipMalloc((void **)&d_length, sizeof(float)));
    CUDA_CHECK(
        hipMemcpy(d_length, &length, sizeof(float), hipMemcpyHostToDevice)
    );

    d_epsilon = nullptr;
    CUDA_CHECK(hipMalloc((void **)&d_epsilon, sizeof(float)));
    CUDA_CHECK(
        hipMemcpy(d_epsilon, &epsilon, sizeof(float), hipMemcpyHostToDevice)
    );

    gridSize =
        (inputSize.first * inputSize.second + BLOCK_SIZE - 1) / BLOCK_SIZE;
}

void BatchNorm2d::delCUDA() {
    hipFree(d_output);
    hipFree(d_running_mean);
    hipFree(d_running_var);
    hipFree(d_weights);
    hipFree(d_biases);
    hipFree(d_length);
    hipFree(d_epsilon);
}

void BatchNorm2d::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_weights, weights.data(), sizeof(float) * inputChannels,
        hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        d_biases, biases.data(), sizeof(float) * inputChannels,
        hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        d_running_mean, running_mean.data(), sizeof(float) * inputChannels,
        hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        d_running_var, running_var.data(), sizeof(float) * inputChannels,
        hipMemcpyHostToDevice
    ));
}

float *BatchNorm2d::forwardCUDA(const float *d_input) {
    // Compute per-channel batch normalization
    for (int i = 0; i < inputChannels; i++) {
        // Subtract mean from input
        Kernels::vec_scalar_sub<<<gridSize, BLOCK_SIZE>>>(
            d_input + i * inputSize.first * inputSize.second,
            d_output + i * inputSize.first * inputSize.second,
            &d_running_mean[i], inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());

        // Divide by sqrt(running_var + epsilon)
        Kernels::vec_scale<<<gridSize, BLOCK_SIZE>>>(
            d_output + i * inputSize.first * inputSize.second,
            d_output + i * inputSize.first * inputSize.second,
            &d_running_var[i], d_epsilon, inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());

        // Multiply by weights
        Kernels::vec_scalar_mul<<<gridSize, BLOCK_SIZE>>>(
            d_output + i * inputSize.first * inputSize.second,
            d_output + i * inputSize.first * inputSize.second, &d_weights[i],
            inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());

        // Add biases
        Kernels::vec_scalar_add<<<gridSize, BLOCK_SIZE>>>(
            d_output + i * inputSize.first * inputSize.second,
            d_output + i * inputSize.first * inputSize.second, &d_biases[i],
            inputSize.first * inputSize.second
        );
        CUDA_CHECK(hipGetLastError());
    }

    activation->activate(d_output);

    return d_output;
}