#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "vector.cuh"
#include "matmul.cuh"
#include "cuda_helper.cuh"

using namespace CUDANet;

void Utils::print_vec(float* d_vec, const unsigned int length) {
    std::vector<float> h_vec(length);
    CUDA_CHECK(hipMemcpy(
        h_vec.data(), d_vec, sizeof(float) * length, hipMemcpyDeviceToHost
    ));

    for (int i = 0; i < length; ++i) {
        std::cout << h_vec[i] << ", ";
    }

    std::cout << std::endl;
}

void Utils::clear(float* d_vec, const unsigned int length) {
    CUDA_CHECK(hipMemset(d_vec, 0, sizeof(float) * length));
}

void Utils::max(float* d_vec, float* d_max, const unsigned int length) {
    
    const int grid_size = (length + BLOCK_SIZE - 1) / BLOCK_SIZE;
    Kernels::max_reduce<<<grid_size, BLOCK_SIZE>>>(d_vec, d_max, length);
    CUDA_CHECK(hipGetLastError());

    int remaining = grid_size;
    while (remaining > 1) {
        int blocks_needed = (remaining + BLOCK_SIZE - 1) / BLOCK_SIZE;
        CUDANet::Kernels::max_reduce<<<blocks_needed, BLOCK_SIZE>>>(d_max, d_max, remaining);
        CUDA_CHECK(hipGetLastError());

        remaining = blocks_needed;
    }

}

void Utils::sum(float* d_vec, float* d_sum, const unsigned int length) {
    
    const int gridSize = (length + BLOCK_SIZE - 1) / BLOCK_SIZE;

    CUDANet::Kernels::sum_reduce<<<gridSize, BLOCK_SIZE>>>(
        d_vec, d_sum, length
    );
    CUDA_CHECK(hipGetLastError());

    int remaining = gridSize;
    while (remaining > 1) {
        int blocks_needed = (remaining + BLOCK_SIZE - 1) / BLOCK_SIZE;
        CUDANet::Kernels::sum_reduce<<<blocks_needed, BLOCK_SIZE>>>(d_sum, d_sum, remaining);
        CUDA_CHECK(hipGetLastError());

        remaining = blocks_needed;
    }
}