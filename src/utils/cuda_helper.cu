#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>

#include "cuda_helper.cuh"

hipDeviceProp_t initializeCUDA() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        std::fprintf(stderr, "No CUDA devices found. Exiting.\n");
        std::exit(EXIT_FAILURE);
    }

    int device = 0;
    CUDA_CHECK(hipSetDevice(device));

    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, device));

    std::printf("Using CUDA device %d: %s\n", device, deviceProp.name);

    return deviceProp;
}