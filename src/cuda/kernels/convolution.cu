#include "hip/hip_runtime.h"
#include <iostream>

#include "convolution.cuh"

using namespace CUDANet;

__global__ void Kernels::convolution(
    const float* __restrict__ d_input,
    const float* __restrict__ d_kernel,
    const float* __restrict__ d_bias,
    float* __restrict__ d_output,
    const shape2d inputSize,
    const int   nChannels,
    const shape2d paddingSize,
    const shape2d kernelSize,
    const shape2d stride,
    const int   nFilters,
    const shape2d outputSize
) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int f = blockDim.z * blockIdx.z + threadIdx.z;

    if (i >= outputSize.first || j >= outputSize.second || f >= nFilters) {
        return;
    }

    float sum = 0.0f;

    // Iterate over kernel and input matrix
    for (int c = 0; c < nChannels; c++) {
        for (int k = 0; k < kernelSize.first; k++) {
            for (int l = 0; l < kernelSize.second; l++) {
                // if i, j is in the padding region
                if (i * stride.first + k < paddingSize.first ||
                    i * stride.first + k >=
                        (inputSize.first + paddingSize.first) ||
                    j * stride.second + l < paddingSize.second ||
                    j * stride.second + l >=
                        (inputSize.second + paddingSize.second)) {
                    continue;
                }

                int kernelIndex =
                    f * kernelSize.first * kernelSize.second * nChannels +
                    c * kernelSize.first * kernelSize.second +
                    k * kernelSize.second + l;
                int inputIndex = c * inputSize.first * inputSize.second +
                                 (i * stride.first + k - paddingSize.first) *
                                     inputSize.second +
                                 (j * stride.second + l - paddingSize.second);

                sum += d_kernel[kernelIndex] * d_input[inputIndex];
            }
        }
    }

    d_output[f * outputSize.first * outputSize.second + i * outputSize.second + j] =
        sum + d_bias[f];
}