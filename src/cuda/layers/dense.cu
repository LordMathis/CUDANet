#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <functional>
#include <iostream>

#include "vector.cuh"
#include "activation.cuh"
#include "cuda_helper.cuh"
#include "dense.cuh"
#include "matmul.cuh"

using namespace CUDANet::Layers;

Dense::Dense(
    int                inputSize,
    int                outputSize,
    ActivationType activationType
)
    : inputSize(inputSize), outputSize(outputSize) {
    // Allocate memory for weights and biases
    weights.resize(outputSize * inputSize);
    biases.resize(outputSize);

    initializeWeights();
    initializeBiases();

    d_output = nullptr;

    CUDA_CHECK(hipMalloc((void**)&d_output, sizeof(float) * outputSize));

    d_weights = nullptr;
    d_biases  = nullptr;

    // Allocate GPU memory for weights and biases
    CUDA_CHECK(
        hipMalloc((void**)&d_weights, sizeof(float) * inputSize * outputSize)
    );
    CUDA_CHECK(hipMalloc((void**)&d_biases, sizeof(float) * outputSize));
    toCuda();

    // Calculate block and grid sizes
    forwardGridSize =
        (std::max(inputSize, outputSize) + BLOCK_SIZE - 1) / BLOCK_SIZE;
    biasGridSize = (outputSize + BLOCK_SIZE - 1) / BLOCK_SIZE;

    activation = new Activation(activationType, outputSize);
}

Dense::~Dense() {
    hipFree(d_output);
    hipFree(d_weights);
    hipFree(d_biases);
    delete activation;
}

void Dense::initializeWeights() {
    std::fill(weights.begin(), weights.end(), 0.0f);
}

void Dense::initializeBiases() {
    std::fill(biases.begin(), biases.end(), 0.0f);
}

float* Dense::forward(const float* d_input) {

    Kernels::mat_vec_mul<<<forwardGridSize, BLOCK_SIZE>>>(
        d_weights, d_input, d_output, inputSize, outputSize
    );
    CUDA_CHECK(hipGetLastError());

    Kernels::vec_vec_add<<<biasGridSize, BLOCK_SIZE>>>(
        d_biases, d_output, d_output, outputSize
    );
    CUDA_CHECK(hipGetLastError());

    activation->activate(d_output);
    CUDA_CHECK(hipDeviceSynchronize());

    return d_output;
}

void Dense::toCuda() {
    CUDA_CHECK(hipMemcpy(
        d_weights, weights.data(), sizeof(float) * inputSize * outputSize,
        hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        d_biases, biases.data(), sizeof(float) * outputSize,
        hipMemcpyHostToDevice
    ));
}

void Dense::setWeights(const float* weights_input) {
    std::copy(weights_input, weights_input + weights.size(), weights.begin());
    toCuda();
}

std::vector<float> Dense::getWeights() {
    return weights;
}

void Dense::setBiases(const float* biases_input) {
    std::copy(biases_input, biases_input + biases.size(), biases.begin());
    toCuda();
}

std::vector<float> Dense::getBiases() {
    return biases;
}

int Dense::getOutputSize() {
    return outputSize;
}

int Dense::getInputSize() {
    return inputSize;
}