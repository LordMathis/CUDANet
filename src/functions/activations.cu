#include "hip/hip_runtime.h"
#include <functional>

#include "activations.cuh"

__device__ float sigmoid(float a) {
    return 1.0 / (1.0 + exp(-a));
}

__device__ float relu(float a) {
    return a < 0.0 ? 0.0 : a;
}

__device__ float linear(float a) {
    return a;
}

__global__ void sigmoid_kernel(
    const float* __restrict__ src,
    float* __restrict__ dst,
    int len
) {
    int stride = gridDim.x * blockDim.x;
    int tid    = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = tid; i < len; i += stride) {
        dst[i] = sigmoid(src[i]);
    }
}

__global__ void
relu_kernel(const float* __restrict__ src, float* __restrict__ dst, int len) {
    int stride = gridDim.x * blockDim.x;
    int tid    = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = tid; i < len; i += stride) {
        dst[i] = relu(src[i]);
    }
}

__global__ void
linear_kernel(const float* __restrict__ src, float* __restrict__ dst, int len) {
    int stride = gridDim.x * blockDim.x;
    int tid    = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = tid; i < len; i += stride) {
        dst[i] = linear(src[i]);
    }
}
