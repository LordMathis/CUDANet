#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "matmul.cuh"

using namespace CUDANet;

__global__ void Kernels::mat_vec_mul(
    const float* __restrict__ d_matrix,
    const float* __restrict__ d_vector,
    float* __restrict__ d_output,
    const unsigned int w,
    const unsigned int h
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < h) {
        float temp = 0.0f;

        for (unsigned int j = 0; j < w; j++) {
            temp += d_matrix[tid * w + j] * d_vector[j];
        }

        d_output[tid] = temp;
    }
}

__global__ void Kernels::vec_vec_add(
    const float* __restrict__ d_vector1,
    const float* __restrict__ d_vector2,
    float* __restrict__ d_output,
    const unsigned int w
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= w) {
        return;
    }
    d_output[tid] = d_vector1[tid] + d_vector2[tid];
}

__global__ void Kernels::vec_scalar_sub(
    const float* __restrict__ d_src,
    float* __restrict__ d_out,
    const float* __restrict__ d_scalar,
    const unsigned int len
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= len) {
        return;
    }
    d_out[tid] = d_src[tid] - *d_scalar;
}

__global__ void Kernels::vec_scalar_div(
    const float* __restrict__ d_src,
    float* __restrict__ d_out,
    const float* __restrict__ d_scalar,
    const unsigned int len
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= len) {
        return;
    }
    d_out[tid] = d_src[tid] / *d_scalar;
}

__global__ void Kernels::vec_exp(
    const float* __restrict__ src,
    float* __restrict__ dst,
    const unsigned int len
) {
    int stride = gridDim.x * blockDim.x;
    int tid    = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = tid; i < len; i += stride) {
        dst[i] = expf(src[i]);
    }
}


__global__ void Kernels::max_reduce(
    const float* __restrict__ d_vector,
    float* __restrict__ d_output,
    const unsigned int len
) {
    __shared__ float shared_max[BLOCK_SIZE];
    int i       = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len) {
        shared_max[threadIdx.x] = d_vector[i];
    } else {
        shared_max[threadIdx.x] = -INFINITY;
    }    

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shared_max[threadIdx.x] = fmaxf(shared_max[threadIdx.x], shared_max[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        d_output[blockIdx.x] = shared_max[0];
    }
}

__global__ void Kernels::sum_reduce(
    const float* __restrict__ d_vector,
    float* __restrict__ d_output,
    const unsigned int len
) {
    __shared__ float partial_sum[BLOCK_SIZE];
    int              i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len) {
        partial_sum[threadIdx.x] = d_vector[i];
    } else {
        partial_sum[threadIdx.x] = 0.0f;
    }

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        d_output[blockIdx.x] = partial_sum[0];
    }
}