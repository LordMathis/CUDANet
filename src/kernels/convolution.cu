#include "hip/hip_runtime.h"
#include <iostream>

#include "convolution.cuh"

/*
Pads matrix width x height x n_channels to width + 2 * padding x height + 2 *
padding x n_channels Matrix is represented as a pointer to a vector

For example:

w = 2
h = 3
n = 2
p = 1

Channel 0:
  0  1
  2  3
  4  5
Channel 1:
  6  7
  8  9
 10 11

Is represented as:

0 1 2 3 4 5 6 7 8 9 10 11

Padded result (as a continuous vector):

0.0f, 0.0f, 0.0f, 0.0f,
0.0f, 0.0f, 1.0f, 0.0f,
0.0f, 2.0f, 3.0f, 0.0f,
0.0f, 4.0f, 5.0f, 0.0f,
0.0f, 0.0f, 0.0f, 0.0f,
0.0f, 0.0f, 0.0f, 0.0f,
0.0f, 6.0f, 7.0f, 0.0f,
0.0f, 8.0f, 9.0f, 0.0f,
9.0f, 10.0f, 11.0f, 0.0f,
0.0f, 0.0f, 0.0f, 0.0f

Args:
  d_input: Pointer to input vector representing matrix
  d_padded: Pointer to output vector representing padded matrix (needs to be
pre-allocated)
  w: Width of input matrix
  h: Height of input matrix
  n: Number of channels in input matrix
  p: Padding
*/
__global__ void CUDANet::Kernels::padding(
    const float* __restrict__ d_input,
    float* __restrict__ d_padded,
    const unsigned int w,
    const unsigned int h,
    const unsigned int n,
    const unsigned int p
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= (w + 2 * p) * (h + 2 * p) * n) {
        return;
    }

    int idx = tid;

    // unravel index into padded matrix
    int i_n = idx / ((w + 2 * p) * (h + 2 * p));
    int i_h = idx % ((w + 2 * p) * (h + 2 * p)) / (w + 2 * p);
    int i_w = idx % (w + 2 * p);

    // if i is in the padding region
    if (i_w < p || i_w >= (w + p) || i_h < p || i_h >= (h + p)) {
        d_padded[tid] = 0.0f;
    } else {
        // Get index into input vector
        int i_orig    = i_n * w * h + (i_h - p) * w + (i_w - p);
        d_padded[tid] = d_input[i_orig];
    }
}

__global__ void CUDANet::Kernels::convolution(
    const float* __restrict__ d_input,
    const float* __restrict__ d_kernel,
    float* __restrict__ d_output,
    const int inputSize,
    const int nChannels,
    const int paddingSize,
    const int kernelSize,
    const int stride,
    const int nFilters,
    const int outputSize
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= outputSize * outputSize * nFilters) {
        return;
    }

    // Get output index
    int f = tid / (outputSize * outputSize);
    int i = tid % (outputSize * outputSize) / outputSize;
    int j = tid % outputSize;

    float sum = 0.0f;

    // Iterate over kernel and input matrix
    for (int c = 0; c < nChannels; c++) {
        for (int k = 0; k < kernelSize; k++) {
            for (int l = 0; l < kernelSize; l++) {
                int kernelIndex = f * kernelSize * kernelSize * nChannels +
                                  c * kernelSize * kernelSize + k * kernelSize +
                                  l;
                int inputIndex = c * inputSize * inputSize +
                                 (i * stride + k) * inputSize +
                                 (j * stride + l);

                sum += d_kernel[kernelIndex] * d_input[inputIndex];
            }
        }
    }

    d_output[tid] = sum;
}