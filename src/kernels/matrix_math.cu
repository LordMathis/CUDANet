#include "hip/hip_runtime.h"
#include "matrix_math.cuh"

__global__ void mat_vec_mul_kernel(
    const float* d_matrix,
    const float* d_vector,
    float*       d_output,
    int          w,
    int          h
) {
    
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= w * h) {
        return;
    }

    for (int i = 0; i < w; i++) {
        d_output[tid] += d_matrix[tid * w + i] * d_vector[i];
    }

}

__global__ void vec_vec_add_kernel(
    const float* d_vector1,
    const float* d_vector2,
    float*       d_output,
    int          w
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= w) {
        return;
    }
    d_output[tid] = d_vector1[tid] + d_vector2[tid];
}
