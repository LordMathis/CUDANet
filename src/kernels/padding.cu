
#include <hip/hip_runtime.h>
#include <vector>

/*
Pads matrix width x height x n_channels to width + 2 * padding x height + 2 *
padding x n_channels Matrix is represented as a pointer to a vector

For example:

w = 2
h = 3
n = 2
p = 1

Channel 0:
  0  1
  2  3
  4  5
Channel 1:
  6  7
  8  9
 10 11

Is represented as:

0 1 2 3 4 5 6 7 8 9 10 11

Padded result (as a continuous vector):

0.0f, 0.0f, 0.0f, 0.0f,
0.0f, 0.0f, 1.0f, 0.0f,
0.0f, 2.0f, 3.0f, 0.0f,
0.0f, 4.0f, 5.0f, 0.0f,
0.0f, 0.0f, 0.0f, 0.0f,
0.0f, 0.0f, 0.0f, 0.0f,
0.0f, 6.0f, 7.0f, 0.0f,
0.0f, 8.0f, 9.0f, 0.0f,
9.0f, 10.0f, 11.0f, 0.0f,
0.0f, 0.0f, 0.0f, 0.0f

Args:
  d_input: Pointer to input vector representing matrix
  d_padded: Pointer to output vector representing padded matrix (needs to be
pre-allocated)
  w: Width of input matrix
  h: Height of input matrix
  n: Number of channels in input matrix
  p: Padding
*/
__global__ void pad_matrix_kernel(
    const float* d_input,
    float*       d_padded,
    int          w,
    int          h,
    int          n,
    int          p
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= (w + 2 * p) * (h + 2 * p) * n) {
        return;
    }

    int idx = tid;

    // unravel index into padded matrix
    int i_n = idx / ((w + 2 * p) * (h + 2 * p));
    int i_h = idx % ((w + 2 * p) * (h + 2 * p)) / (w + 2 * p);
    int i_w = idx % (w + 2 * p);

    // if i is in the padding region
    if (i_w < p || i_w >= (w + p) || i_h < p || i_h >= (h + p)) {
        d_padded[tid] = 0.0f;
    } else {
        // Get index into input vector
        int i_orig    = i_n * w * h + (i_h - p) * w + (i_w - p);
        d_padded[tid] = d_input[i_orig];
    }
}
