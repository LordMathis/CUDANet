
#include <hip/hip_runtime.h>
/*
Pads matrix width x height x n_channels to width + 2 * padding x height + 2 *
padding x n_channels Matrix is represented as a pointer to column major vector

For example:

w = 2
h = 3
n = 2
p = 1

Channel 0:
  0  1
  2  3
  4  5
Channel 1:
  6  7
  8  9
 10 11

Is represented as:

0 2 4 1 3 5 6 8 10 7 9 11

Padded result:

0 0 0 0 0 0 0 2 4 0 0 1 3 5 0 0 0 0 0 0 0 0 0 0 0 0 6 8 10 0 0 7 9 11 0 0 0 0 0 0


Args:
  d_input: Pointer to input vector representing matrix
  d_padded: Pointer to output vector representing padded matrix (needs to be
pre-allocated) w: Width of input matrix h: Height of input matrix n: Number of
channels in input matrix p: Padding
*/
__global__ void pad_matrix_kernel(
    const float* d_input,
    float*       d_padded,
    int          w,
    int          h,
    int          n,
    int          p
) {
    int stride = gridDim.x * blockDim.x;
    int tid    = blockDim.x * blockIdx.x + threadIdx.x;

	for (int i = tid; i < (w + 2 * p) * (h + 2 * p) * n; i += stride) {
		
		// if i is in the padding region
		if (i < p * (h + 2 * p) * n || i >= (w + p) * (h + 2 * p) * n) {
			d_padded[i] = 0.0f;
		} else {
			// if i is in the original region
			d_padded[i] = d_input[(i - p * (h + 2 * p) * n) / (h + 2 * p) * w + (i - p * (h + 2 * p) * n) % (h + 2 * p)];
		}
	}
}